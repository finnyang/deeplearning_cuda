#include "DL/util/net.h"
#include <stdlib.h>
#include <string.h>

 void make_convolution(struct Net* net, char* name, int bottom_size, char** bottom_name, int top_size, char** top_name,
		int num_output, int pad, int stride, int kernel_size, int dilation){
	init();
	new_layer->type = CONVOLUTION;
	struct ConvolutionParameter* convolution_param = &(new_layer->parameter.convolution_param);
	convolution_param->kernel_size = kernel_size;
	convolution_param->num_output = num_output;
	convolution_param->pad = pad;
	convolution_param->stride = stride;
	convolution_param->dilation = dilation;
	new_layer->has_learn_parameter = 2;
	new_layer->learn_parameter = (struct Blob**)malloc(sizeof(struct Blob*)*2);
	new_layer->learn_parameter[0] = (struct Blob*)malloc(sizeof(struct Blob));
	new_layer->learn_parameter[1] = (struct Blob*)malloc(sizeof(struct Blob));
	if(pad == 0 && dilation == 1 && stride == 1){
		new_layer->med_num = 1;
		new_layer->meds = (struct Blob**)malloc(sizeof(struct Blob*));
		new_layer->meds[0] = (struct Blob*)malloc(sizeof(struct Blob));
	}else{
		new_layer->med_num = 2;
		new_layer->meds = (struct Blob**)malloc(sizeof(struct Blob*)*2);
		new_layer->meds[0] = (struct Blob*)malloc(sizeof(struct Blob));
		new_layer->meds[1] = (struct Blob*)malloc(sizeof(struct Blob));
	}
}

void make_input(struct Net* net, char* name, int top_size, char** top_name,
		int n, int c, int h, int w){
    int i;
	net->layer_num += 1;
	net->tops_num += top_size;
	net->layer_parameter = (struct LayerParameter*)realloc(net->layer_parameter, sizeof(struct LayerParameter)*net->layer_num);
	struct LayerParameter* new_layer = net->layer_parameter+(net->layer_num-1);
	new_layer->name = name;
    new_layer->bottom_num = 0;
	new_layer->top_num = top_size;
	new_layer->bottom = NULL;
	new_layer->top = (char**)malloc(sizeof(char*)*top_size);
	new_layer->bottoms = NULL;
	new_layer->tops = (struct Blob**)malloc(sizeof(struct Blob*)*top_size);
	net->tops = (struct Blob**)realloc(net->tops, sizeof(struct Blob*)*net->tops_num);
	net->names = (char**)realloc(net->names, sizeof(char*)*net->tops_num);
	for(i = 0; i < top_size; ++i){
	  net->names[net->tops_num-top_size+i] = top_name[i];
	  new_layer->top[i] = top_name[i];
	  net->tops[net->tops_num-top_size+i] = (struct Blob*)malloc(sizeof(struct Blob));
	  new_layer->tops[i]=(net->tops[net->tops_num-top_size+i]);
	}
	new_layer->type = INPUT;
	struct InputParameter* input_param = &(new_layer->parameter.input_param);
	input_param->n = n;
	input_param->c = c;
	input_param->h = h;
	input_param->w = w;
	new_layer->has_learn_parameter = 0;
	new_layer->learn_parameter = NULL;
	new_layer->med_num = 0;
	new_layer->meds = NULL;
}

void make_relu(struct Net* net, char* name, int bottom_size, char** bottom_name, int top_size, char** top_name,
		float negative_slope){
	init();
	new_layer->type = RELU;
	struct ReluParameter* relu_param = &(new_layer->parameter.relu_param);
	relu_param->negative_slope = negative_slope;
	new_layer->has_learn_parameter = 0;
	new_layer->learn_parameter = NULL;
	new_layer->med_num = 0;
	new_layer->meds = NULL;
}

void make_pooling(struct Net* net, char* name, int bottom_size, char** bottom_name, int top_size, char** top_name,
		int pad, int stride, int kernel_size, enum PoolingType pooling_type, int global_pooling){
    init();
	new_layer->type = POOLING;
	struct PoolingParameter* pooling_param = &(new_layer->parameter.pooling_param);
	pooling_param->kernel_size = kernel_size;
	pooling_param->pad = pad;
	pooling_param->stride = stride;
	pooling_param->pooling_type = pooling_type;
	pooling_param->global_pooling = global_pooling;
	if(global_pooling){
		pooling_param->pad = 0;
		pooling_param->stride = 1;
	}
	new_layer->has_learn_parameter = 0;
	new_layer->learn_parameter = NULL;
	new_layer->med_num = 0;
	new_layer->meds = NULL;
}

void make_innerproduct(struct Net* net, char* name, int bottom_size, char** bottom_name, int top_size, char** top_name,
		int num_output){
    init();
	new_layer->type = INNERPRODUCT;
	struct InnerproductParameter* innerproduct_param = &(new_layer->parameter.innerproduct_param);
	innerproduct_param->num_output = num_output;
	new_layer->has_learn_parameter = 2;
	new_layer->learn_parameter = (struct Blob**)malloc(sizeof(struct Blob*)*2);
	new_layer->learn_parameter[0] = (struct Blob*)malloc(sizeof(struct Blob));
	new_layer->learn_parameter[1] = (struct Blob*)malloc(sizeof(struct Blob));
	new_layer->med_num = 1;
	new_layer->meds =(struct Blob**)malloc(sizeof(struct Blob*));
	new_layer->meds[0] = (struct Blob*)malloc(sizeof(struct Blob));
//	new_layer->med_num = 0;
//	new_layer->meds = NULL;
}

void make_softmax(struct Net* net, char* name, int bottom_size, char** bottom_name, int top_size, char**top_name,
		int axis){
	init();
	new_layer->type = SOFTMAX;
	struct SoftmaxParameter* softmax_param = &(new_layer->parameter.softmax_param);//根据层特有的参数来确定
	softmax_param->axis = axis;
	new_layer->has_learn_parameter = 0;
	new_layer->learn_parameter = NULL;
	new_layer->med_num = 1;
	new_layer->meds =(struct Blob**)malloc(sizeof(struct Blob*));
	new_layer->meds[0] = (struct Blob*)malloc(sizeof(struct Blob));
}

void make_permute(struct Net* net, char* name, int bottom_size, char** bottom_name, int top_size, char** top_name,
		int idx_n, int idx_c, int idx_h, int idx_w){
	init();
	new_layer->type = PERMUTE;
	struct PermuteParameter* permute_param = &(new_layer->parameter.permute_param);
	permute_param->idx_n = idx_n;
	permute_param->idx_c = idx_c;
	permute_param->idx_h = idx_h;
	permute_param->idx_w = idx_w;
	new_layer->has_learn_parameter = 0;
	new_layer->learn_parameter = NULL;
	new_layer->med_num = 0;
	new_layer->meds = NULL;
}

void make_flatten(struct Net* net, char* name, int bottom_size, char** bottom_name, int top_size, char** top_name,
		int start_axis, int end_axis){
	init();
	new_layer->type = FLATTEN;
	struct FlattenParameter* flatten_param = &(new_layer->parameter.flatten_param);;
	flatten_param->start_axis = start_axis;
	flatten_param->end_axis = end_axis;
	new_layer->has_learn_parameter = 0;
	new_layer->learn_parameter = NULL;
	new_layer->med_num = 0;
	new_layer->meds = NULL;
}

void make_reshape(struct Net* net, char* name, int bottom_size, char** bottom_name, int top_size, char** top_name,
		int dim_n, int dim_c, int dim_h, int dim_w){
    init();
	new_layer->type = RESHAPE;
	struct ReshapeParameter* reshape_param = &(new_layer->parameter.reshape_param);
	reshape_param->dim_n = dim_n;
	reshape_param->dim_c = dim_c;
	reshape_param->dim_h = dim_h;
	reshape_param->dim_w = dim_w;
	new_layer->has_learn_parameter = 0;
	new_layer->learn_parameter = NULL;
	new_layer->med_num = 0;
	new_layer->meds = NULL;
}

void make_concat(struct Net* net, char* name, int bottom_size, char** bottom_name, int top_size, char** top_name,
		int axis){
	init();
	new_layer->type = CONCAT;
	struct ConcatParameter* concat_param = &(new_layer->parameter.concat_param);
	concat_param->axis = axis;
	new_layer->has_learn_parameter = 0;
	new_layer->learn_parameter = NULL;
	new_layer->med_num = 0;
	new_layer->meds = NULL;
}

void make_priorbox(struct Net* net, char* name, int bottom_size, char** bottom_name, int top_size, char** top_name,
		int min_size, int max_size, float aspect1, float aspect2, float v1, float v2, float v3, float v4){
	init();
	new_layer->type = PRIORBOX;
	struct PriorboxParameter* priorbox_param = &(new_layer->parameter.priorbox_param);
	priorbox_param->min_size = min_size;
	priorbox_param->max_size = max_size;
	priorbox_param->variance[0] = v1;
	priorbox_param->variance[1] = v2;
	priorbox_param->variance[2] = v3;
	priorbox_param->variance[3] = v4;
	i = 3;
	j = 2;
	priorbox_param->aspect[0] = aspect1;
	priorbox_param->aspect[1] = 1/aspect1;
	if(aspect2 > 0){
		priorbox_param->aspect[2] = aspect2;
		priorbox_param->aspect[3] = 1/aspect2;
		i+=2;
		j+=2;
	}
	if(max_size > 0){
		++i;
	}
	priorbox_param->priorbox_num = i;
	priorbox_param->aspect_num = j;
	new_layer->has_learn_parameter = 0;
	new_layer->learn_parameter = NULL;
	new_layer->med_num = 0;
	new_layer->meds = NULL;
}

void make_normalize(struct Net* net, char* name, int bottom_size, char** bottom_name, int top_size, char** top_name,
		int channel_shared, int across_spatial) {
	init();
	new_layer->type = NORMALIZE;
	struct NormalizeParameter* normalize_param = &(new_layer->parameter.normalize_param);
	normalize_param->across_spatial = across_spatial;
	normalize_param->channel_shared = channel_shared;
	new_layer->has_learn_parameter = 1;
	new_layer->learn_parameter = (struct Blob**)malloc(sizeof(struct Blob*));
	new_layer->learn_parameter[0] = (struct Blob*)malloc(sizeof(struct Blob));
	new_layer->med_num = 0;
	new_layer->meds = NULL;
}

void set_net_io(struct Net* net, int input_size, char** input_name, int output_size, char** output_name){
	int i,j;
	net->input_num = input_size;
	net->output_num = output_size;
	net->inputs = (struct Blob**)malloc(sizeof(struct Blob*)*input_size);
	net->outputs = (struct Blob**)malloc(sizeof(struct Blob*)*output_size);
	for(i = 0; i < input_size; ++i){
		for(j = 0; j < net->tops_num; ++j){
			if(strcmp(input_name[i], net->names[j]) == 0){
				net->inputs[i] = net->tops[j];
			}
		}
	}
	for(i = 0; i < output_size; ++i){
		for(j = 0; j < net->tops_num; ++j){
			if(strcmp(output_name[i], net->names[j]) == 0){
				net->outputs[i] = net->tops[j];
			}
		}
	}
}

//void InitLayerName(struct LayerName* layername){
//	//num表示当前层的个数
//	int num = 3;
//	int i = 0;
//    layername->names = NULL;
//	layername->names = (char**)realloc(layername->names, sizeof(char*)*num);
//	layername->names[i++] = "input";
//	layername->names[i++] = "relu";
//    layername->names[i++] = "flatten";
//}

//void DeleteLayerName(struct LayerName* layername){
//	free(layername->names);
//}

void forward(struct Net net){
  int i = 0;
  for(i = 0; i < net.layer_num; ++i){
    net.handle.functions[net.layer_parameter[i].type].forward_cpu(net.layer_parameter+i);
  }
}

void setup(struct Net net) {
  int i = 0;
  for(i = 0; i < net.layer_num; ++i){
    net.handle.functions[net.layer_parameter[i].type].setup(net.layer_parameter+i);
  }
}

void deletenet(struct Net net){//网络的内存释放
	int i, j;
	hipblasDestroy(net.cublas_handle);
	free(net.inputs);
	free(net.outputs);
	for (i = 0; i < net.tops_num; ++i){
		FreeBlob(net.tops[i]);
		free(net.tops[i]);
	}
	free(net.tops);
	free(net.names);
	for(i = 0; i < net.layer_num; ++i){
		free(net.layer_parameter[i].bottom);
		free(net.layer_parameter[i].top);
		free(net.layer_parameter[i].bottoms);
		free(net.layer_parameter[i].tops);
		if(net.layer_parameter[i].has_learn_parameter){
			for(j = 0; j < net.layer_parameter[i].has_learn_parameter; ++j){
				FreeBlob(net.layer_parameter[i].learn_parameter[j]);
				free(net.layer_parameter[i].learn_parameter[j]);
			}
			free(net.layer_parameter[i].learn_parameter);
		}
		if(net.layer_parameter[i].med_num){
#ifdef DEBUG
			printf("realease %s\n", net.layer_parameter[i].name);
#endif
			for(j = 0; j < net.layer_parameter[i].med_num; ++j){
				FreeBlob(net.layer_parameter[i].meds[j]);
				free(net.layer_parameter[i].meds[j]);
			}
			free(net.layer_parameter[i].meds);
		}
	}
	free(net.layer_parameter);
	free(net.handle.functions);
}
/*

void make_innerproduct(struct Net* net, char* name, int bottom_size, char** bottom_name, int top_size, char** top_name,
		int num_output){
    init();
	new_layer->type = INNERPRODUCT;
	struct InnerproductParameter* innerproduct_param = &(new_layer->parameter.innerproduct_param);
	innerproduct_param->num_output = num_output;
	new_layer->has_learn_parameter = 2;
	new_layer->learn_parameter = (struct Blob**)malloc(sizeof(struct Blob*)*2);
	new_layer->learn_parameter[0] = (struct Blob*)malloc(sizeof(struct Blob));
	new_layer->learn_parameter[1] = (struct Blob*)malloc(sizeof(struct Blob));
}

void make_softmax(struct Net* net, char* name, int bottom_size, char** bottom_name, int top_size, char**top_name,
		int axis){
	init();
	new_layer->type = SOFTMAX;
	struct SoftmaxParameter* softmax_param = &(new_layer->parameter.softmax_param);//根据层特有的参数来确定
	softmax_param->axis = axis;
	new_layer->has_learn_parameter = 0;
	new_layer->learn_parameter = NULL;
}



void make_permute(struct Net* net, char* name, int bottom_size, char** bottom_name, int top_size, char** top_name,
		int idx_n, int idx_c, int idx_h, int idx_w){
	init();
	new_layer->type = PERMUTE;
	struct PermuteParameter* permute_param = &(new_layer->parameter.permute_param);
	permute_param->idx_n = idx_n;
	permute_param->idx_c = idx_c;
	permute_param->idx_h = idx_h;
	permute_param->idx_w = idx_w;
	new_layer->has_learn_parameter = 0;
	new_layer->learn_parameter = NULL;
}

void make_concat(struct Net* net, char* name, int bottom_size, char** bottom_name, int top_size, char** top_name,
		int axis){
	init();
	new_layer->type = CONCAT;
	struct ConcatParameter* concat_param = &(new_layer->parameter.concat_param);
	concat_param->axis = axis;
	new_layer->has_learn_parameter = 0;
	new_layer->learn_parameter = NULL;
}

void make_priorbox(struct Net* net, char* name, int bottom_size, char** bottom_name, int top_size, char** top_name,
		int min_size, int max_size, float aspect1, float aspect2, float v1, float v2, float v3, float v4){
	init();
	new_layer->type = PRIORBOX;
	struct PriorboxParameter* priorbox_param = &(new_layer->parameter.priorbox_param);
	priorbox_param->min_size = min_size;
	priorbox_param->max_size = max_size;
	priorbox_param->variance[0] = v1;
	priorbox_param->variance[1] = v2;
	priorbox_param->variance[2] = v3;
	priorbox_param->variance[3] = v4;
	i = 3;
	j = 2;
	priorbox_param->aspect[0] = aspect1;
	priorbox_param->aspect[1] = 1/aspect1;
	if(aspect2 > 0){
		priorbox_param->aspect[2] = aspect2;
		priorbox_param->aspect[3] = 1/aspect2;
		i+=2;
		j+=2;
	}
	if(max_size > 0){
		++i;
	}
	priorbox_param->priorbox_num = i;
	priorbox_param->aspect_num = j;
	new_layer->has_learn_parameter = 0;
	new_layer->learn_parameter = NULL;
}

void make_normalize(struct Net* net, char* name, int bottom_size, char** bottom_name, int top_size, char** top_name,
		int channel_shared, int across_spatial) {
	init();
	new_layer->type = NORMALIZE;
	struct NormalizeParameter* normalize_param = &(new_layer->parameter.normalize_param);
	normalize_param->across_spatial = across_spatial;
	normalize_param->channel_shared = channel_shared;
	new_layer->has_learn_parameter = 1;
	new_layer->learn_parameter = (struct Blob**)malloc(sizeof(struct Blob*)*1);
	new_layer->learn_parameter[0] = (struct Blob*)malloc(sizeof(struct Blob));
}
 */
