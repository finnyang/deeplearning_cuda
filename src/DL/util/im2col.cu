#include "hip/hip_runtime.h"
#include "DL/util/im2col.h"
#include "DL/util/layer.h"
//void im2col_cpu(float* data_im,
//		int channels, int height, int width,
//		int ksize, int stride, int pad, int dilation, float* data_col){
//    int c,h,w;
//    int height_col = (height - (dilation*(ksize-1)+1) + 2*pad) / stride + 1;
//    int width_col = (width - (dilation*(ksize-1)+1) + 2*pad) / stride + 1;
//    int channels_col = channels * ksize * ksize;
//    for(c = 0; c < channels_col; ++c){
//        int w_offset = c % ksize;
//        int h_offset = (c / ksize) % ksize;
//        int c_im = c / ksize / ksize;
//        for(h = 0; h < height_col; ++h){
//        	for(w = 0; w < width_col; ++w){
//        		int im_row = h_offset*dilation+h*stride-pad;
//        		int im_col = w_offset*dilation+w*stride-pad;
//        		int col_index = (c * height_col + h) * width_col + w;
//				if( im_row < 0 || im_col < 0 || im_row > height-1 || im_col > width-1)
//					data_col[col_index] = 0.0;
//				else{
//					data_col[col_index] = data_im[(c_im*height+im_row)*width+im_col];
//				}
//        	}
//        }
//    }
//}
/*__global__ void im2col_gpu_kernel(float* data_im,
		int channels, int height, int width,
		int ksize, int stride, int pad, int dilation, float* data_col, int height_col, int width_col, int channels_col){
	int idx_idx = blockDim.x*blockIdx.x+threadIdx.x;
	if(idx_idx < channels_col){
		int w_offset = idx_idx % ksize;
		int h_offset = (idx_idx / ksize) % ksize;
		int c_im = idx_idx / ksize / ksize;
		int w, h;
		for(h = 0; h < height_col; ++h){
			for(w = 0; w < width_col; ++w){
				int im_row = h_offset*dilation+h*stride-pad;
				int im_col = w_offset*dilation+w*stride-pad;
				int col_index = (idx_idx * height_col + h) * width_col + w;
				if( im_row < 0 || im_col < 0 || im_row > height-1 || im_col > width-1)
					data_col[col_index] = 0.0;
				else{
					data_col[col_index] = data_im[(c_im*height+im_row)*width+im_col];
				}
			}
		}
	}else{
		return;
	}

}*/
__global__ void im2col_gpu_kernel(float* data_im,
		int channels, int height, int width,
		int ksize, int stride, int pad, int dilation, float* data_col, int height_col, int width_col, int channels_col){
//	int idx_idx = blockDim.x*blockIdx.x+threadIdx.x;
//	if(idx_idx < channels_col){
//		int w_offset = idx_idx % ksize;
//		int h_offset = (idx_idx / ksize) % ksize;
//		int c_im = idx_idx / ksize / ksize;
//		int w, h;
//		for(h = 0; h < height_col; ++h){
//			for(w = 0; w < width_col; ++w){
//				int im_row = h_offset*dilation+h*stride-pad;
//				int im_col = w_offset*dilation+w*stride-pad;
//				int col_index = (idx_idx * height_col + h) * width_col + w;
//				if( im_row < 0 || im_col < 0 || im_row > height-1 || im_col > width-1)
//					data_col[col_index] = 0.0;
//				else{
//					data_col[col_index] = data_im[(c_im*height+im_row)*width+im_col];
//				}
//			}
//		}
//	}else{
//		return;
//	}

	int idx = blockDim.x*blockIdx.x+threadIdx.x;
	if(idx < height_col*width_col*channels_col){
		int idx_idx = idx/(height_col*width_col);
		int temp = idx%(height_col*width_col);
		int w_offset = idx_idx % ksize;
		int h_offset = (idx_idx / ksize) % ksize;
		int c_im = idx_idx / ksize / ksize;
		int h = temp/width_col;
		int w = temp%width_col;
		int im_row = h_offset*dilation+h*stride-pad;
		int im_col = w_offset*dilation+w*stride-pad;
		int col_index = (idx_idx * height_col + h) * width_col + w;
		if( im_row < 0 || im_col < 0 || im_row > height-1 || im_col > width-1)
			data_col[col_index] = 0.0;
		else{
			data_col[col_index] = data_im[(c_im*height+im_row)*width+im_col];
		}
	}
}

__global__ void im2col_gpu_kernel_new(float* data_im,
		int channels, int height, int width,
		int ksize, int stride, int pad, int dilation, float* data_col, int height_col, int width_col, int channels_col){
	int w_offset = blockIdx.y;
	int h_offset = blockIdx.x;
	int c_im = blockIdx.z;
	int h = threadIdx.x;
	int w = threadIdx.y;

	if(w_offset < ksize && h_offset < ksize && c_im < channels && h < height_col && w < width_col){
		int idx_idx = ((c_im*ksize)+h_offset)*ksize+w_offset;
		int im_row = h_offset*dilation+h*stride-pad;
		int im_col = w_offset*dilation+w*stride-pad;
		int col_index = (idx_idx * height_col + h) * width_col + w;
		if( im_row < 0 || im_col < 0 || im_row > height-1 || im_col > width-1)
			data_col[col_index] = 0.0;
		else{
			data_col[col_index] = data_im[(c_im*height+im_row)*width+im_col];
		}
	}
}

void im2col_gpu(float* data_im,
		int channels, int height, int width,
		int ksize, int stride, int pad, int dilation, float* data_col){
    int height_col = (height - (dilation*(ksize-1)+1) + 2*pad) / stride + 1;
    int width_col = (width - (dilation*(ksize-1)+1) + 2*pad) / stride + 1;
    int channels_col = channels * ksize * ksize;
    im2col_gpu_kernel<<<(channels_col*height_col*width_col+CUDA_NUM_THREADS-1)/CUDA_NUM_THREADS, CUDA_NUM_THREADS>>>(data_im,
    		channels, height, width,
    		ksize, stride, pad, dilation, data_col, height_col, width_col, channels_col);
//        dim3 grid((ksize+CUDA_NUM_THREADS-1)/CUDA_NUM_THREADS,
//        		(ksize+CUDA_NUM_THREADS-1)/CUDA_NUM_THREADS,
//        		(channels+CUDA_NUM_THREADS-1)/CUDA_NUM_THREADS);
//        dim3 block((height_col+CUDA_NUM_THREADS-1)/CUDA_NUM_THREADS,
//        		(width_col+CUDA_NUM_THREADS-1)/CUDA_NUM_THREADS);
//        im2col_gpu_kernel_new<<<grid, block>>>(data_im,
//            		channels, height, width,
//            		ksize, stride, pad, dilation, data_col, height_col, width_col, channels_col);

}
