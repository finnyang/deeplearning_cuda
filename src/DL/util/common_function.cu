#include "hip/hip_runtime.h"
#include "DL/util/common_function.h"

__global__ void set_data(float* input, int count, float scale){
	int idx = blockDim.x*blockIdx.x+threadIdx.x;
	if(idx < count)
		input[idx] = scale;
}

void set(float* input, int count, float scale){
	set_data<<<(count+CUDA_NUM_THREADS-1)/CUDA_NUM_THREADS, CUDA_NUM_THREADS>>>(input, count, scale);
}

__global__ void copy_data(float* input, float* output, int count){
	int idx = blockDim.x*blockIdx.x+threadIdx.x;
	if(idx < count){
		output[idx] = input[idx];
	}
}

void copy(float* input, float* output, int count){
	copy_data<<<(count+CUDA_NUM_THREADS-1)/CUDA_NUM_THREADS, CUDA_NUM_THREADS>>>(input, output, count);
}
__global__ void exp_data(float* input, float* output, int count){
	int idx = blockDim.x*blockIdx.x+threadIdx.x;
	if(idx < count){
		output[idx] = exp(input[idx]);
	}
}

void exp_gpu(float* input, float* output, int count){
	exp_data<<<(count+CUDA_NUM_THREADS-1)/CUDA_NUM_THREADS, CUDA_NUM_THREADS>>>(input, output, count);
}

__global__ void power_data(float* input, float* output, int count){
	int idx = blockDim.x*blockIdx.x+threadIdx.x;
	if(idx < count){
		output[idx] = input[idx]*input[idx];
	}
}

void power_gpu(float* input, float* output, int count){
	power_data<<<(count+CUDA_NUM_THREADS-1)/CUDA_NUM_THREADS, CUDA_NUM_THREADS>>>(input, output, count);
}

__global__ void scale_data(float* input, float* output, int count, float scale){
	int idx = blockDim.x*blockIdx.x+threadIdx.x;
	if(idx < count){
		output[idx] = input[idx]*scale;
	}
}

void scale_gpu(float* input, float* output, int count, float scale){
	scale_data<<<(count+CUDA_NUM_THREADS-1)/CUDA_NUM_THREADS, CUDA_NUM_THREADS>>>(input, output, count, scale);
}
