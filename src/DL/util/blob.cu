#include "DL/util/blob.h"
#include <malloc.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

void MakeBlob(int n, int c, int h, int w, struct Blob* blob){
	blob->n = n;
	blob->c = c;
	blob->h = h;
	blob->w = w;
	blob->own = 1;
	blob->count = n*c*h*w;
	hipMalloc(&(blob->gpu_data), sizeof(float)*blob->count);
}
void FreeBlob(struct Blob* blob){
	if (blob->gpu_data != NULL && blob->own){
		blob->n = 0;
		blob->c = 0;
		blob->h = 0;
		blob->w = 0;
		blob->count = 0;
		hipFree(blob->gpu_data);
	}
}
