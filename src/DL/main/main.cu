
#include <hip/hip_runtime.h>
#ifdef SSD
#include <stdio.h>
#include "DL/util/net.h"
#include "DL/util/layer_factory.h"
#include <cuda_runtime.h>
#include <time.h>

void confignet(struct Net* net){
	  net->layer_num = 0;
	  net->tops_num = 0;
	  net->tops = NULL;
	  net->names = NULL;
	  net->layer_parameter = NULL;

	  char* input[1] = {"data"};
	  make_input(net, "data", 1, input, 1, 3, 448, 448);

	  char* b_1[1] = {"data"};
	  char* t_1[1] = {"conv1"};
	  make_convolution(net, "conv1", 1, b_1, 1, t_1, 64, 3, 2, 7, 1);

	  char* b_2[1] = {"conv1"};
	  char* t_2[1] = {"ReLU1"};
	  make_relu(net, "ReLU1", 1, b_2, 1, t_2, 0.1);

	  char* b_3[1] = {"ReLU1"};
	  char* t_3[1] = {"pool1"};
	  make_pooling(net, "pool1", 1, b_3, 1, t_3, 0, 2, 2, MAX, 0);

	  char* b_4[1] = {"pool1"};
	  char* t_4[1] = {"conv2"};
	  make_convolution(net, "conv2", 1, b_4, 1, t_4, 192, 1, 1, 3, 1);

	  char* b_5[1] = {"conv2"};
	  char* t_5[1] = {"ReLU2"};
	  make_relu(net, "ReLU2", 1, b_5, 1, t_5, 0.1);

	  char* b_6[1] = {"ReLU2"};
	  char* t_6[1] = {"pool2"};
	  make_pooling(net, "pool2", 1, b_6, 1, t_6, 0, 2, 2, MAX, 0);

	  char* b_7[1] = {"pool2"};
	  char* t_7[1] = {"conv3"};
	  make_convolution(net, "conv3", 1, b_7, 1, t_7, 128, 0, 1, 1, 1);

	  char* b_8[1] = {"conv3"};
	  char* t_8[1] = {"ReLU3"};
	  make_relu(net, "ReLU3", 1, b_8, 1, t_8, 0.1);

	  char* b_9[1] = {"ReLU3"};
	  char* t_9[1] = {"conv4"};
	  make_convolution(net, "conv4", 1, b_9, 1, t_9, 256, 1, 1, 3, 1);

	  char* b_10[1] = {"conv4"};
	  char* t_10[1] = {"ReLU4"};
	  make_relu(net, "ReLU3", 1, b_10, 1, t_10, 0.1);

	  char* b_11[1] = {"ReLU4"};
	  char* t_11[1] = {"conv5"};
	  make_convolution(net, "conv4", 1, b_11, 1, t_11, 256, 0, 1, 1, 1);

	  char* b_12[1] = {"conv5"};
	  char* t_12[1] = {"ReLU5"};
	  make_relu(net, "ReLU5", 1, b_12, 1, t_12, 0.1);

	  char* b_13[1] = {"ReLU5"};
	  char* t_13[1] = {"conv6"};
	  make_convolution(net, "conv4", 1, b_13, 1, t_13, 512, 1, 1, 3, 1);

	  char* b_14[1] = {"conv6"};
	  char* t_14[1] = {"ReLU6"};
	  make_relu(net, "ReLU6", 1, b_14, 1, t_14, 0.1);

	  char* b_15[1] = {"ReLU6"};
	  char* t_15[1] = {"pool3"};
	  make_pooling(net, "pool3", 1, b_15, 1, t_15, 0, 2, 2, MAX, 0);

	  char* b_16[1] = {"pool3"};
	  char* t_16[1] = {"conv7"};
	  make_convolution(net, "conv7", 1, b_16, 1, t_16, 256, 0, 1, 1, 1);

	  char* b_17[1] = {"conv7"};
	  char* t_17[1] = {"ReLU7"};
	  make_relu(net, "ReLU7", 1, b_17, 1, t_17, 0.1);

	  char* b_18[1] = {"ReLU7"};
	  char* t_18[1] = {"conv8"};
	  make_convolution(net, "conv8", 1, b_18, 1, t_18, 512, 1, 1, 3, 1);

	  char* b_19[1] = {"conv8"};
	  char* t_19[1] = {"ReLU8"};
	  make_relu(net, "ReLU8", 1, b_19, 1, t_19, 0.1);

	  char* b_20[1] = {"ReLU8"};
	  char* t_20[1] = {"conv9"};
	  make_convolution(net, "conv9", 1, b_20, 1, t_20, 256, 0, 1, 1, 1);

	  char* b_21[1] = {"conv9"};
	  char* t_21[1] = {"ReLU9"};
	  make_relu(net, "ReLU9", 1, b_21, 1, t_21, 0.1);

	  char* b_22[1] ={"ReLU9"};
	  char* t_22[1] = {"conv10"};
	  make_convolution(net, "conv10", 1, b_22, 1, t_22, 512, 1, 1, 3, 1);

	  char* b_23[1] ={"conv10"};
	  char* t_23[1] = {"ReLU10"};
	  make_relu(net, "ReLU10", 1, b_23, 1, t_23, 0.1);

	  char* b_24[1] ={"ReLU10"};
	  char* t_24[1] = {"conv11"};
	  make_convolution(net, "conv11", 1, b_24, 1, t_24, 256, 0, 1, 1, 1);

	  char* b_25[1] ={"conv11"};
	  char* t_25[1] = {"ReLU11"};
	  make_relu(net, "ReLU11", 1, b_25, 1, t_25, 0.1);

	  char* b_26[1] ={"ReLU11"};
	  char* t_26[1] = {"conv12"};
	  make_convolution(net, "conv12", 1, b_26, 1, t_26, 512, 1, 1, 3, 1);

	  char* b_27[1] ={"conv12"};
	  char* t_27[1] = {"ReLU12"};
	  make_relu(net, "ReLU12", 1, b_27, 1, t_27, 0.1);

	  char* b_28[1] ={"ReLU12"};
	  char* t_28[1] = {"conv13"};
	  make_convolution(net, "conv13", 1, b_28, 1, t_28, 256, 0, 1, 1, 1);

	  char* b_29[1] ={"conv13"};
	  char* t_29[1] = {"ReLU13"};
	  make_relu(net, "ReLU13", 1, b_29, 1, t_29, 0.1);

	  char* b_30[1] ={"ReLU13"};
	  char* t_30[1] = {"conv14"};
	  make_convolution(net, "conv14", 1, b_30, 1, t_30, 512, 1, 1, 3, 1);

	  char* b_31[1] ={"conv14"};
	  char* t_31[1] = {"ReLU14"};
	  make_relu(net, "ReLU14", 1, b_31, 1, t_31, 0.1);

	  char* b_32[1] ={"ReLU14"};
	  char* t_32[1] = {"conv15"};
	  make_convolution(net, "conv15", 1, b_32, 1, t_32, 512, 0, 1, 1, 1);

	  char* b_33[1] ={"conv15"};
	  char* t_33[1] = {"ReLU15"};
	  make_relu(net, "ReLU15", 1, b_33, 1, t_33, 0.1);

	  char* b_34[1] ={"ReLU15"};
	  char* t_34[1] = {"conv16"};
	  make_convolution(net, "conv16", 1, b_34, 1, t_34, 1024, 1, 1, 3, 1);

	  char* b_35[1] ={"conv16"};
	  char* t_35[1] = {"ReLU16"};
	  make_relu(net, "ReLU16", 1, b_35, 1, t_35, 0.1);

	  char* b_36[1] = {"ReLU16"};
	  char* t_36[1] = {"pool4"};
	  make_pooling(net, "pool4", 1, b_36, 1, t_36, 0, 2, 2, MAX, 0);

	  char* b_37[1] ={"pool4"};
	  char* t_37[1] = {"conv17"};
	  make_convolution(net, "conv17", 1, b_37, 1, t_37, 512, 0, 1, 1, 1);

	  char* b_38[1] ={"conv17"};
	  char* t_38[1] = {"ReLU17"};
	  make_relu(net, "ReLU17", 1, b_38, 1, t_38, 0.1);

	  char* b_39[1] ={"ReLU17"};
	  char* t_39[1] = {"conv18"};
	  make_convolution(net, "conv18", 1, b_39, 1, t_39, 1024, 1, 1, 3, 1);

	  char* b_40[1] ={"conv18"};
	  char* t_40[1] = {"ReLU18"};
	  make_relu(net, "ReLU18", 1, b_40, 1, t_40, 0.1);

	  char* b_41[1] ={"ReLU18"};
	  char* t_41[1] = {"conv19"};
	  make_convolution(net, "conv19", 1, b_41, 1, t_41, 512, 0, 1, 1, 1);

	  char* b_42[1] ={"conv19"};
	  char* t_42[1] = {"ReLU19"};
	  make_relu(net, "ReLU19", 1, b_42, 1, t_42, 0.1);

	  char* b_43[1] ={"ReLU19"};
	  char* t_43[1] = {"conv20"};
	  make_convolution(net, "conv20", 1, b_43, 1, t_43, 1024, 1, 1, 3, 1);

	  char* b_44[1] ={"conv20"};
	  char* t_44[1] = {"ReLU20"};
	  make_relu(net, "ReLU20", 1, b_44, 1, t_44, 0.1);

	  char* b_45[1] ={"ReLU20"};
	  char* t_45[1] = {"conv21"};
	  make_convolution(net, "conv21", 1, b_45, 1, t_45, 1024, 1, 1, 3, 1);

	  char* b_46[1] ={"conv21"};
	  char* t_46[1] = {"ReLU21"};
	  make_relu(net, "ReLU21", 1, b_46, 1, t_46, 0.1);

	  char* b_47[1] ={"ReLU21"};
	  char* t_47[1] = {"conv22"};
	  make_convolution(net, "conv22", 1, b_47, 1, t_47, 1024, 1, 2, 3, 1);

	  char* b_48[1] ={"conv22"};
	  char* t_48[1] = {"ReLU22"};
	  make_relu(net, "ReLU22", 1, b_48, 1, t_48, 0.1);

	  char* b_49[1] ={"ReLU22"};
	  char* t_49[1] = {"conv23"};
	  make_convolution(net, "conv23", 1, b_49, 1, t_49, 1024, 1, 1, 3, 1);

	  char* b_50[1] ={"conv23"};
	  char* t_50[1] = {"ReLU23"};
	  make_relu(net, "ReLU23", 1, b_50, 1, t_50, 0.1);

	  char* b_51[1] ={"ReLU23"};
	  char* t_51[1] = {"conv24"};
	  make_convolution(net, "conv24", 1, b_51, 1, t_51, 1024, 1, 1, 3, 1);

	  char* b_52[1] ={"conv24"};
	  char* t_52[1] = {"ReLU24"};
	  make_relu(net, "ReLU24", 1, b_52, 1, t_52, 0.1);

	  char* b_53[1] = {"ReLU24"};
	  char* t_53[1] = {"connect1"};
	  make_innerproduct(net, "connect1", 1, b_53, 1, t_53, 4096);

	  char* b_54[1] ={"connect1"};
	  char* t_54[1] = {"ReLU25"};
	  make_relu(net, "ReLU25", 1, b_54, 1, t_54, 0.1);

	  char* b_55[1] = {"ReLU25"};
	  char* t_55[1] = {"connect2"};
	  make_innerproduct(net, "connect2", 1, b_55, 1, t_55, 1470);

	  char* inputblobs[1] = {"data"};
	  char* outputblobs[1] = {"connect2"};
	  set_net_io(net, 1, inputblobs, 1, outputblobs);
}

void configssd(struct Net* net){
	  net->layer_num = 0;
	  net->tops_num = 0;
	  net->tops = NULL;
	  net->names = NULL;
	  net->layer_parameter = NULL;

	  char* input[1] = {"data"};
	  make_input(net, "data", 1, input, 1, 3, 500, 500);

	  char* b_1[1] = {"data"};
	  char* t_1[1] = {"conv1_1"};
	  make_convolution(net, "conv1_1", 1, b_1, 1, t_1, 64, 1, 1, 3, 1);


	  char* b_2[1] = {"conv1_1"};
	  char* t_2[1] = {"conv1_1_relu"};
	  make_relu(net, "relu1_1", 1, b_2, 1, t_2, 0);

	  char* b_3[1] = {"conv1_1_relu"};
	  char* t_3[1] = {"conv1_2"};
	  make_convolution(net, "conv1_2", 1, b_3, 1, t_3, 64, 1, 1, 3, 1);

	  char* b_4[1] = {"conv1_2"};
	  char* t_4[1] = {"conv1_2_relu"};
	  make_relu(net, "relu1_2", 1, b_4, 1, t_4, 0);

	  char* b_5[1] = {"conv1_2_relu"};
	  char* t_5[1] = {"pool1"};
	  make_pooling(net, "pool1", 1, b_5, 1, t_5, 0, 2, 2, MAX, 0);//

	  char* b_6[1] = {"pool1"};
	  char* t_6[1] = {"conv2_1"};
	  make_convolution(net, "conv2_1", 1, b_6, 1, t_6, 128, 1, 1, 3, 1);

	  char* b_7[1] = {"conv2_1"};
	  char* t_7[1] = {"conv2_1_relu"};
	  make_relu(net, "relu2_1", 1, b_7, 1, t_7, 0);

	  char* b_8[1] = {"conv2_1_relu"};
	  char* t_8[1] = {"conv2_2"};
	  make_convolution(net, "conv2_2", 1, b_8, 1, t_8, 128, 1, 1, 3, 1);

	  char* b_9[1] = {"conv2_2"};
	  char* t_9[1] = {"conv2_2_relu"};
	  make_relu(net, "relu2_2", 1, b_9, 1, t_9, 0);

	  char* b_10[1] = {"conv2_2_relu"};
	  char* t_10[1] = {"pool2"};
	  make_pooling(net, "pool2", 1, b_10, 1, t_10, 0, 2, 2, MAX, 0);

	  char* b_11[1] = {"pool2"};
	  char* t_11[1] = {"conv3_1"};
	  make_convolution(net, "conv3_1", 1, b_11, 1, t_11, 256, 1, 1, 3, 1);

	  char* b_12[1] = {"conv3_1"};
	  char* t_12[1] = {"conv3_1_relu"};
	  make_relu(net, "relu3_1", 1, b_12, 1, t_12, 0);

	  char* b_13[1] = {"conv3_1_relu"};
	  char* t_13[1] = {"conv3_2"};
	  make_convolution(net, "conv3_2", 1, b_13, 1, t_13, 256, 1, 1, 3, 1);

	  char* b_14[1] = {"conv3_2"};
	  char* t_14[1] = {"conv3_2_relu"};
	  make_relu(net, "relu3_2", 1, b_14, 1, t_14, 0);

	  char* b_15[1] = {"conv3_2_relu"};
	  char* t_15[1] = {"conv3_3"};
	  make_convolution(net, "conv3_3", 1, b_15, 1, t_15, 256, 1, 1, 3, 1);

	  char* b_16[1] = {"conv3_3"};
	  char* t_16[1] = {"conv3_3_relu"};
	  make_relu(net, "relu3_3", 1, b_16, 1, t_16, 0);

	  char* b_17[1] = {"conv3_3_relu"};
	  char* t_17[1] = {"pool3"};
	  make_pooling(net, "pooling3", 1, b_17, 1, t_17, 0, 2, 2, MAX, 0);

	  char* b_18[1] = {"pool3"};
	  char* t_18[1] = {"conv4_1"};
	  make_convolution(net, "conv4_1", 1, b_18, 1, t_18, 512, 1, 1, 3, 1);

	  char* b_19[1] = {"conv4_1"};
	  char* t_19[1] = {"conv4_1_relu"};
	  make_relu(net, "relu4_1", 1, b_19, 1, t_19, 0);

	  char* b_20[1] = {"conv4_1_relu"};
	  char* t_20[1] = {"conv4_2"};
	  make_convolution(net, "conv4_2", 1, b_20, 1, t_20, 512, 1, 1, 3, 1);

	  char* b_21[1] = {"conv4_2"};
	  char* t_21[1] = {"conv4_2_relu"};
	  make_relu(net, "relu4_2", 1, b_21, 1, t_21, 0);

	  char* b_22[1] = {"conv4_2_relu"};
	  char* t_22[1] = {"conv4_3"};
	  make_convolution(net, "conv4_3", 1, b_22, 1, t_22, 512, 1, 1, 3, 1);

	  char* b_23[1] = {"conv4_3"};
	  char* t_23[1] = {"conv4_3_relu"};
	  make_relu(net, "relu4_3", 1, b_23, 1, t_23, 0);

	  char* b_24[1] = {"conv4_3_relu"};
	  char* t_24[1] = {"pool4"};
	  make_pooling(net, "pool4", 1, b_24, 1, t_24, 0, 2, 2, MAX, 0);

	  char* b_25[1] = {"pool4"};
	  char* t_25[1] = {"conv5_1"};
	  make_convolution(net, "conv5_1", 1, b_25, 1, t_25, 512, 1, 1, 3, 1);

	  char* b_26[1] = {"conv5_1"};
	  char* t_26[1] = {"conv5_1_relu"};
	  make_relu(net, "relu5_1", 1, b_26, 1, t_26, 0);

	  char* b_27[1] = {"conv5_1_relu"};
	  char* t_27[1] = {"conv5_2"};
	  make_convolution(net, "conv5_2", 1, b_27, 1, t_27, 512, 1, 1, 3, 1);

	  char* b_28[1] = {"conv5_2"};
	  char* t_28[1] = {"conv5_2_relu"};
	  make_relu(net, "relu5_2", 1, b_28, 1, t_28, 0);

	  char* b_29[1] = {"conv5_2_relu"};
	  char* t_29[1] = {"conv5_3"};
	  make_convolution(net, "conv5_3", 1, b_29, 1, t_29, 512, 1, 1, 3, 1);

	  char* b_30[1] = {"conv5_3"};
	  char* t_30[1] = {"conv5_3_relu"};
	  make_relu(net, "relu5_3", 1, b_30, 1, t_30, 0);

	  char* b_31[1] = {"conv5_3_relu"};
	  char* t_31[1] = {"pool5"};
	  make_pooling(net, "pool5", 1, b_31, 1, t_31, 1, 1, 3, MAX, 0);

	  char* b_32[1] = {"pool5"};
	  char* t_32[1] = {"fc6"};
	  make_convolution(net, "fc6", 1, b_32, 1, t_32, 1024, 6, 1, 3, 6);

	  char* b_33[1] = {"fc6"};
	  char* t_33[1] = {"fc6_relu"};
	  make_relu(net, "relu6", 1, b_33, 1, t_33, 0);

	  char* b_34[1] = {"fc6_relu"};
	  char* t_34[1] = {"fc7"};
	  make_convolution(net, "fc7", 1, b_34, 1, t_34, 1024, 0, 1, 1, 1);

	  char* b_35[1] = {"fc7"};
	  char* t_35[1] = {"fc7_relu"};
	  make_relu(net, "relu7", 1, b_35, 1, t_35, 0);

	  char* b_36[1] = {"fc7_relu"};
	  char* t_36[1] = {"conv6_1"};
	  make_convolution(net, "conv6_1", 1, b_36, 1, t_36, 256, 0, 1, 1, 1);

	  char* b_37[1] = {"conv6_1"};
	  char* t_37[1] = {"conv6_1_relu"};
	  make_relu(net, "conv6_1_relu", 1, b_37, 1, t_37, 0);

	  char* b_38[1] = {"conv6_1_relu"};
	  char* t_38[1] = {"conv6_2"};
	  make_convolution(net, "conv6_2", 1, b_38, 1, t_38, 512, 1, 2, 3, 1);

	  char* b_39[1] = {"conv6_2"};
	  char* t_39[1] = {"conv6_2_relu"};
	  make_relu(net, "conv6_2_relu", 1, b_39, 1, t_39, 0);

	  char* b_40[1] = {"conv6_2_relu"};
	  char* t_40[1] = {"conv7_1"};
	  make_convolution(net, "conv7_1", 1, b_40, 1, t_40, 128, 0, 1, 1, 1);

	  char* b_41[1] = {"conv7_1"};
	  char* t_41[1] = {"conv7_1_relu"};
	  make_relu(net, "conv7_1_relu", 1, b_41, 1, t_41, 0);

	  char* b_42[1] = {"conv7_1_relu"};
	  char* t_42[1] = {"conv7_2"};
	  make_convolution(net, "conv7_2", 1, b_42, 1, t_42, 256, 1, 2, 3, 1);

	  char* b_43[1] = {"conv7_2"};
	  char* t_43[1] = {"conv7_2_relu"};
	  make_relu(net, "conv7_2_relu", 1, b_43, 1, t_43, 0);

	  char* b_44[1] = {"conv7_2_relu"};
	  char* t_44[1] = {"conv8_1"};
	  make_convolution(net, "conv8_1", 1, b_44, 1, t_44, 128, 0, 1, 1, 1);

	  char* b_45[1] = {"conv8_1"};
	  char* t_45[1] = {"conv8_1_relu"};
	  make_relu(net, "conv8_2_relu", 1, b_45, 1, t_45, 0);

	  char* b_46[1] = {"conv8_1_relu"};
	  char* t_46[1] = {"conv8_2"};
	  make_convolution(net, "conv8_2", 1, b_46, 1, t_46, 256, 1, 2, 3, 1);

	  char* b_47[1] = {"conv8_2"};
	  char* t_47[1] = {"conv8_2_relu"};
	  make_relu(net, "conv8_2_relu", 1, b_47, 1, t_47, 0);

	  char* b_48[1] = {"conv8_2_relu"};
	  char* t_48[1] = {"conv9_1"};
	  make_convolution(net, "conv9_1", 1, b_48, 1, t_48, 128, 0, 1, 1, 1);

	  char* b_49[1] = {"conv9_1"};
	  char* t_49[1] = {"conv9_1_relu"};
	  make_relu(net, "conv9_1", 1, b_49, 1, t_49, 0);

	  char* b_50[1] = {"conv9_1_relu"};
	  char* t_50[1] = {"conv9_2"};
	  make_convolution(net, "conv9_2", 1, b_50, 1, t_50, 256, 1, 2, 3, 1);

	  char* b_51[1] = {"conv9_2"};
	  char* t_51[1] = {"conv9_2_relu"};
	  make_relu(net, "conv9_2_relu", 1, b_51, 1, t_51, 0);

	  char* b_52[1] = {"conv9_2_relu"};
	  char* t_52[1] = {"pool6"};
	  make_pooling(net, "pool6", 1, b_52, 1, t_52, 0, 0, 0, AVE, 1);

	  char* b_53[1] = {"conv4_3_relu"};
	  char* t_53[1] = {"conv4_3_norm"};
	  make_normalize(net, "conv4_3_norm", 1, b_53, 1, t_53, 0, 0);

	  char* b_54[1] = {"conv4_3_norm"};
	  char* t_54[1] = {"conv4_3_norm_mbox_loc"};
	  make_convolution(net, "conv4_3_norm_mbox_loc", 1, b_54, 1, t_54, 12, 1, 1, 3, 1);

	  char* b_55[1] = {"conv4_3_norm_mbox_loc"};
	  char* t_55[1] = {"conv4_3_norm_mbox_loc_perm"};
	  make_permute(net, "conv4_3_norm_mbox_loc_perm", 1, b_55, 1, t_55, 0, 2, 3, 1);

	  char* b_56[1] = {"conv4_3_norm_mbox_loc_perm"};
	  char* t_56[1] = {"conv4_3_norm_mbox_loc_flat"};
	  make_flatten(net, "conv4_3_norm_mbox_loc_flat", 1, b_56, 1, t_56, 1, 3);

	  char* b_57[1] = {"conv4_3_norm"};
	  char* t_57[1] = {"conv4_3_norm_mbox_conf"};
	  make_convolution(net, "conv4_3_norm_mbox_conf", 1, b_57, 1, t_57, 63, 1, 1, 3, 1);

	  char* b_58[1] = {"conv4_3_norm_mbox_conf"};
	  char* t_58[1] = {"conv4_3_norm_mbox_conf_perm"};
	  make_permute(net, "conv4_3_norm_mbox_conf_perm", 1, b_58, 1, t_58, 0, 2, 3, 1);

	  char* b_59[1] = {"conv4_3_norm_mbox_conf_perm"};
	  char* t_59[1] = {"conv4_3_norm_mbox_conf_flat"};
	  make_flatten(net, "conv4_3_norm_mbox_conf_flat", 1, b_59, 1, t_59, 1, 3);

	  char* b_60[2] = {"conv4_3_norm","data"};
	  char* t_60[1] = {"conv4_3_norm_mbox_priorbox"};
	  make_priorbox(net, "conv4_3_norm_mbox_priorbox", 2, b_60, 1, t_60, 35, -1, 2.0, -1.0, 0.1, 0.1, 0.2, 0.2);

	  char* b_61[1] = {"fc7_relu"};
	  char* t_61[1] = {"fc7_mbox_loc"};
	  make_convolution(net, "fc7_mbox_loc", 1, b_61, 1, t_61, 24, 1, 1, 3, 1);

	  char* b_62[1] = {"fc7_mbox_loc"};
	  char* t_62[1] = {"fc7_mbox_loc_perm"};
	  make_permute(net, "fc7_mbox_loc_perm", 1, b_62, 1, t_62, 0, 2, 3, 1);

	  char* b_63[1] = {"fc7_mbox_loc_perm"};
	  char* t_63[1] = {"fc7_mbox_loc_flat"};
	  make_flatten(net, "fc7_mbox_loc_flat", 1, b_63, 1, t_63, 1, 3);

	  char* b_64[1] = {"fc7_relu"};
	  char* t_64[1] = {"fc7_mbox_conf"};
	  make_convolution(net, "fc7_mbox_conf", 1, b_64, 1, t_64, 126, 1, 1, 3, 1);

	  char* b_65[1] = {"fc7_mbox_conf"};
	  char* t_65[1] = {"fc7_mbox_conf_perm"};
	  make_permute(net, "fc7_mbox_conf_perm", 1, b_65, 1, t_65, 0, 2, 3, 1);

	  char* b_66[1] = {"fc7_mbox_conf_perm"};
	  char* t_66[1] = {"fc7_mbox_conf_flat"};
	  make_flatten(net, "fc7_mbox_conf_flat", 1, b_66, 1, t_66, 1, 3);

	  char* b_67[2] = {"fc7_relu", "data"};
	  char* t_67[1] = {"fc7_mbox_priorbox"};
	  make_priorbox(net, "fc7_mbox_priorbox", 2, b_67, 1, t_67, 75.0, 155.0, 2, 3, 0.1, 0.1, 0.2, 0.2);

	  char* b_68[1] = {"conv6_2_relu"};
	  char* t_68[1] = {"conv6_2_mbox_loc"};
	  make_convolution(net, "conv6_2_mbox_loc", 1, b_68, 1, t_68, 24, 1, 1, 3, 1);

	  char* b_69[1] = {"conv6_2_mbox_loc"};
	  char* t_69[1] = {"conv6_2_mbox_loc_perm"};
	  make_permute(net, "conv6_2_mbox_loc_perm", 1, b_69, 1, t_69, 0, 2, 3, 1);

	  char* b_70[1] = {"conv6_2_mbox_loc_perm"};
	  char* t_70[1] = {"conv6_2_mbox_loc_flat"};
	  make_flatten(net, "conv6_2_mbox_loc_flat", 1, b_70, 1, t_70, 1, 3);

	  char* b_71[1] = {"conv6_2_relu"};
	  char* t_71[1] = {"conv6_2_mbox_conf"};
	  make_convolution(net, "conv6_2_mbox_conf", 1, b_71, 1, t_71, 126, 1, 1, 3, 1);

	  char* b_72[1] = {"conv6_2_mbox_conf"};
	  char* t_72[1] = {"conv6_2_mbox_conf_perm"};
	  make_permute(net, "conv6_2_mbox_conf_perm", 1, b_72, 1, t_72, 0, 2, 3, 1);

	  char* b_73[1] = {"conv6_2_mbox_conf_perm"};
	  char* t_73[1] = {"conv6_2_mbox_conf_flat"};
	  make_flatten(net, "conv6_2_mbox_conf_flat", 1, b_73, 1, t_73, 1, 3);

	  char* b_74[2] = {"conv6_2_relu", "data"};
	  char* t_74[1] = {"conv6_2_mbox_priorbox"};
	  make_priorbox(net, "conv6_2_mbox_priorbox", 2, b_74, 1, t_74, 155.0, 235.0, 2, 3, 0.1, 0.1, 0.2, 0.2);

	  char* b_75[1] = {"conv7_2_relu"};
	  char* t_75[1] = {"conv7_2_mbox_loc"};
	  make_convolution(net, "conv7_2_mbox_loc", 1, b_75, 1, t_75, 24, 1, 1, 3, 1);

	  char* b_76[1] = {"conv7_2_mbox_loc"};
	  char* t_76[1] = {"conv7_2_mbox_loc_perm"};
	  make_permute(net, "conv7_2_mbox_loc_perm", 1, b_76, 1, t_76, 0, 2, 3, 1);

	  char* b_77[1] = {"conv7_2_mbox_loc_perm"};
	  char* t_77[1] = {"conv7_2_mbox_loc_flat"};
	  make_flatten(net, "conv7_2_mbox_loc_flat", 1, b_77, 1, t_77, 1, 3);

	  char* b_78[1] = {"conv7_2_relu"};
	  char* t_78[1] = {"conv7_2_mbox_conf"};
	  make_convolution(net, "conv7_2_mbox_conf", 1, b_78, 1, t_78, 126, 1, 1, 3, 1);

	  char* b_79[1] = {"conv7_2_mbox_conf"};
	  char* t_79[1] = {"conv7_2_mbox_conf_perm"};
	  make_permute(net, "conv7_2_mbox_conf_perm", 1, b_79, 1, t_79, 0, 2, 3, 1);

	  char* b_80[1] = {"conv7_2_mbox_conf_perm"};
	  char* t_80[1] = {"conv7_2_mbox_conf_flat"};
	  make_flatten(net, "conv7_2_mbox_conf_flat", 1, b_80, 1, t_80, 1, 3);

	  char* b_81[2] = {"conv7_2_relu", "data"};
	  char* t_81[1] = {"conv7_2_mbox_priorbox"};
	  make_priorbox(net, "conv7_2_mbox_priorbox", 2, b_81, 1, t_81, 235, 315, 2, 3, 0.1, 0.1, 0.2, 0.2);

	  char* b_82[1] = {"conv8_2_relu"};
	  char* t_82[1] = {"conv8_2_mbox_loc"};
	  make_convolution(net, "conv8_2_mbox_loc", 1, b_82, 1, t_82, 24, 1, 1, 3, 1);

	  char* b_83[1] = {"conv8_2_mbox_loc"};
	  char* t_83[1] = {"conv8_2_mbox_loc_perm"};
	  make_permute(net, "conv8_2_mbox_loc_perm", 1, b_83, 1, t_83, 0, 2, 3, 1);

	  char* b_84[1] = {"conv8_2_mbox_loc_perm"};
	  char* t_84[1] = {"conv8_2_mbox_loc_flat"};
	  make_flatten(net, "conv8_2_mbox_loc_flat", 1, b_84, 1, t_84, 1, 3);

	  char* b_85[1] = {"conv8_2_relu"};
	  char* t_85[1] = {"conv8_2_mbox_conf"};
	  make_convolution(net, "conv8_2_mbox_conf", 1, b_85, 1, t_85, 126, 1, 1, 3, 1);

	  char* b_86[1] = {"conv8_2_mbox_conf"};
	  char* t_86[1] = {"conv8_2_mbox_conf_perm"};
	  make_permute(net, "conv8_2_mbox_conf_perm", 1, b_86, 1, t_86, 0, 2, 3, 1);

	  char* b_87[1] = {"conv8_2_mbox_conf_perm"};
	  char* t_87[1] = {"conv8_2_mbox_conf_flat"};
	  make_flatten(net, "conv8_2_mbox_conf_flat", 1, b_87, 1, t_87, 1, 3);

	  char* b_88[2] = {"conv8_2_relu", "data"};
	  char* t_88[1] = {"conv8_2_mbox_priorbox"};
	  make_priorbox(net, "conv8_2_mbox_priorbox", 2, b_88, 1, t_88, 315, 395, 2, 3, 0.1, 0.1, 0.2, 0.2);

	  char* b_89[1] = {"conv9_2_relu"};
	  char* t_89[1] = {"conv9_2_mbox_loc"};
	  make_convolution(net, "conv9_2_mbox_loc", 1, b_89, 1, t_89, 24, 1, 1, 3, 1);

	  char* b_90[1] = {"conv9_2_mbox_loc"};
	  char* t_90[1] = {"conv9_2_mbox_loc_perm"};
	  make_permute(net, "conv9_2_mbox_loc_perm", 1, b_90, 1, t_90, 0, 2, 3, 1);

	  char* b_91[1] = {"conv9_2_mbox_loc_perm"};
	  char* t_91[1] = {"conv9_2_mbox_loc_flat"};
	  make_flatten(net, "conv9_2_mbox_loc_flat", 1, b_91, 1, t_91, 1, 3);

	  char* b_92[1] = {"conv9_2_relu"};
	  char* t_92[1] = {"conv9_2_mbox_conf"};
	  make_convolution(net, "conv9_2_mbox_conf", 1, b_92, 1, t_92, 126, 1, 1, 3, 1);

	  char* b_93[1] = {"conv9_2_mbox_conf"};
	  char* t_93[1] = {"conv9_2_mbox_conf_perm"};
	  make_permute(net, "conv9_2_mbox_conf_perm", 1, b_93, 1, t_93, 0, 2, 3, 1);

	  char* b_94[1] = {"conv9_2_mbox_conf_perm"};
	  char* t_94[1] = {"conv9_2_mbox_conf_flat"};
	  make_flatten(net, "conv9_2_mbox_conf_flat", 1, b_94, 1, t_94, 1, 3);

	  char* b_95[2] = {"conv9_2_relu", "data"};
	  char* t_95[1] = {"conv9_2_mbox_priorbox"};
	  make_priorbox(net, "conv9_2_mbox_priorbox", 2, b_95, 1, t_95, 395, 475, 2, 3, 0.1, 0.1, 0.2, 0.2);

	  char* b_96[1] = {"pool6"};
	  char* t_96[1] = {"pool6_mbox_loc"};
	  make_convolution(net, "pool6_mbox_loc", 1, b_96, 1, t_96, 24, 1, 1, 3, 1);

	  char* b_97[1] = {"pool6_mbox_loc"};
	  char* t_97[1] = {"pool6_mbox_loc_perm"};
	  make_permute(net, "pool6_mbox_loc_perm", 1, b_97, 1, t_97, 0, 2, 3, 1);

	  char* b_98[1] = {"pool6_mbox_loc_perm"};
	  char* t_98[1] = {"pool6_mbox_loc_flat"};
	  make_flatten(net, "pool6_mbox_loc_flat", 1, b_98, 1, t_98, 1, 3);

	  char* b_99[1]= {"pool6"};
	  char* t_99[1]= {"pool6_mbox_conf"};
	  make_convolution(net, "pool6_mbox_conf", 1, b_99, 1, t_99, 126, 1, 1, 3, 1);

	  char* b_100[1] = {"pool6_mbox_conf"};
	  char* t_100[1] = {"pool6_mbox_conf_perm"};
	  make_permute(net, "pool6_mbox_conf_perm", 1, b_100, 1, t_100, 0, 2, 3, 1);

	  char* b_101[1] = {"pool6_mbox_conf_perm"};
	  char* t_101[1] = {"pool6_mbox_conf_flat"};
	  make_flatten(net, "pool6_mbox_conf_flat", 1, b_101, 1, t_101, 1, 3);

	  char* b_102[2] = {"pool6", "data"};
	  char* t_102[1] = {"pool6_mbox_priorbox"};
	  make_priorbox(net, "pool6_mbox_priorbox", 2, b_102, 1, t_102, 475, 555, 2, 3, 0.1, 0.1, 0.2, 0.2);

	  char* b_103[7] = {"conv4_3_norm_mbox_loc_flat", "fc7_mbox_loc_flat", "conv6_2_mbox_loc_flat", "conv7_2_mbox_loc_flat", "conv8_2_mbox_loc_flat" ,"conv9_2_mbox_loc_flat", "pool6_mbox_loc_flat"};
	  char* t_103[1] = {"mbox_loc"};
	  make_concat(net, "mbox_loc", 7, b_103, 1, t_103, 1);

	  char* b_104[7] = {"conv4_3_norm_mbox_conf_flat", "fc7_mbox_conf_flat", "conv6_2_mbox_conf_flat", "conv7_2_mbox_conf_flat", "conv8_2_mbox_conf_flat" ,"conv9_2_mbox_conf_flat", "pool6_mbox_conf_flat"};
	  char* t_104[1] = {"mbox_conf"};
	  make_concat(net, "mbox_conf", 7, b_104, 1, t_104, 1);

	  char* b_105[7] = {"conv4_3_norm_mbox_priorbox", "fc7_mbox_priorbox", "conv6_2_mbox_priorbox", "conv7_2_mbox_priorbox", "conv8_2_mbox_priorbox" ,"conv9_2_mbox_priorbox", "pool6_mbox_priorbox"};
	  char* t_105[1] = {"mbox_priorbox"};
	  make_concat(net, "mbox_priorbox", 7, b_105, 1, t_105, 2);

	  char* b_106[1] = {"mbox_conf"};
	  char* t_106[1] = {"mbox_conf_reshape"};
	  make_reshape(net, "mbox_conf_reshape", 1, b_106, 1, t_106, 0, -1, 21, 0);

	  char* b_107[1] = {"mbox_conf_reshape"};
	  char* t_107[1] = {"mbox_conf_softmax"};
	  make_softmax(net, "mbox_conf_softmax", 1, b_107, 1, t_107, 2);

	  char* b_108[1] = {"mbox_conf_softmax"};
	  char* t_108[1] = {"mbox_conf_flatten"};
	  make_flatten(net, "mbox_conf_flatten", 1, b_108, 1, t_108, 1, 3);

	  char* inputblobs[1] = {"data"};
	  char* outputblobs[3] = {"mbox_conf_flatten" ,"mbox_loc", "mbox_priorbox"};
	  //char* outputblobs[2] = {"mbox_conf_flatten" ,"mbox_loc"};
	  //char* outputblobs[1] = {"conv4_3_norm_mbox_loc_perm"};
	  //char* outputblobs[2] = {"conv4_3_norm", "pool6"};
	  set_net_io(net, 1, inputblobs, 3, outputblobs);
}
int main(){
  int i;
  double start, finish;
  struct Net net;
  cublasStatus_t status;
  status = cublasCreate(&(net.cublas_handle));
  if(status != CUBLAS_STATUS_SUCCESS){
	  printf("get cublas handle error!\n");
  }else{
	  printf("get cublas handle success!\n");
  }
  //struct LayerName names;
  //InitLayerName(&names);
  struct Map_Type_Setup_Forward handle;
  net.handle.num = 0;
  net.handle.functions = NULL;
  regist_layer(&(net.handle));
  configssd(&net);
  //confignet(&net);
  setup(net);
    printf("network input and ouput\n");
    for(i = 0; i < net.input_num; ++i)
  	  printf("input%d (%d,%d,%d,%d)\n", i+1, net.inputs[i]->n, net.inputs[i]->c, net.inputs[i]->h, net.inputs[i]->w);
    for(i = 0; i < net.output_num; ++i)
  	  printf("output%d (%d,%d,%d,%d)\n", i+1, net.outputs[i]->n, net.outputs[i]->c, net.outputs[i]->h, net.outputs[i]->w);
//  ssd
   char weightpath[200];
   FILE* fp = NULL;
   float* data;
   int j;
   for(i = 0; i < net.layer_num; ++i){
 	  struct LayerParameter param = net.layer_parameter[i];
 	  if(param.has_learn_parameter){
 		  sprintf(weightpath, "%s/%s.txt","./fold_for_test/weights", param.name);
 		  fp = fopen(weightpath, "r");
 		  if(fp == NULL){
 			  printf("%s load error!\n", weightpath);
 			  for(;;);
 		  }else{
 			  for(j = 0; j < param.has_learn_parameter; ++j){
 				  data = (float*)malloc(sizeof(float)*net.layer_parameter[i].learn_parameter[j]->count);
 				  fread(data, sizeof(float), net.layer_parameter[i].learn_parameter[j]->count ,fp);
 				  cudaMemcpy(net.layer_parameter[i].learn_parameter[j]->gpu_data, data, sizeof(float)*net.layer_parameter[i].learn_parameter[j]->count, cudaMemcpyHostToDevice);
 				  free(data);
 			  }
 			  fclose(fp);
 			  printf("%s has loaded!\n", weightpath);
 		  }
//
 	  }
   }
//ssd
   data = (float*)malloc(sizeof(float)*500*3*500);
   for(i = 0; i < 500*500*3; ++i){
 	  data[i] = i%31;
   }
   cudaMemcpy(net.inputs[0]->gpu_data, data, sizeof(float)*500*500*3, cudaMemcpyHostToDevice);

   printf("start count time\n");
	start = clock();
	forward(net);//这个函数代替下面的注释语句
	finish = clock();
	printf( "%f seconds\n",(finish - start) / CLOCKS_PER_SEC);
   fp = NULL;
     char path[200];
     int count = 0;
     for(i = 0; i < net.output_num; ++i){
   	  printf("output %d\n", i);
   	  sprintf(path, "%s/output%d.txt","./fold_for_test/outputs",i);
   	  fp = fopen(path, "r");
   	  count = net.outputs[i]->count;
   	  data = (float*)malloc(sizeof(float)*count);
   	  fread(data, sizeof(float), count, fp);
   	  fclose(fp);
   	  float* data_gpu = (float*)malloc(sizeof(float)*count);
   	  cudaMemcpy(data_gpu, net.outputs[i]->gpu_data, (sizeof(float)*count), cudaMemcpyDeviceToHost);

   	  for(j = 0; j < count; ++j){
   		  if(data[j] - data_gpu[j] > 0.001)
   			  printf("out %d %d %f %f %f\n",i, j, data[j], data_gpu[j], data[j] - data_gpu[j]);
   	  }
   	  free(data_gpu);
   	  free(data);
     }
     printf("network input and ouput\n");
     for(i = 0; i < net.input_num; ++i)
   	  printf("input%d (%d,%d,%d,%d)\n", i+1, net.inputs[i]->n, net.inputs[i]->c, net.inputs[i]->h, net.inputs[i]->w);
     for(i = 0; i < net.output_num; ++i)
   	  printf("output%d (%d,%d,%d,%d)\n", i+1, net.outputs[i]->n, net.outputs[i]->c, net.outputs[i]->h, net.outputs[i]->w);




//  printf("network input and ouput\n");
//  for(i = 0; i < net.input_num; ++i)
//	  printf("input%d (%d,%d,%d,%d)\n", i+1, net.inputs[i]->n, net.inputs[i]->c, net.inputs[i]->h, net.inputs[i]->w);
//  for(i = 0; i < net.output_num; ++i)
//	  printf("output%d (%d,%d,%d,%d)\n", i+1, net.outputs[i]->n, net.outputs[i]->c, net.outputs[i]->h, net.outputs[i]->w);
//  printf("\n");
//  float temp[750000];
//  for(i = 0; i < net.inputs[0]->count; ++i){
//	  temp[i] = i%4;//i%20-10;
//  }
//  cudaMemcpy(net.inputs[0]->gpu_data, temp, sizeof(float)*net.inputs[0]->count, cudaMemcpyHostToDevice);
//  float w_temp[3] = {1,1,1};
//  float b_temp=0.5;
//  cudaMemcpy(net.layer_parameter[4].learn_parameter[0]->gpu_data, w_temp, sizeof(float)*3, cudaMemcpyHostToDevice);
//  cudaMemcpy(net.layer_parameter[4].learn_parameter[1]->gpu_data, &b_temp, sizeof(float), cudaMemcpyHostToDevice);
//    forward(net);
//  float temp1[750000];
//  cudaMemcpy(temp1, net.outputs[0]->gpu_data, sizeof(float)*net.outputs[0]->count, cudaMemcpyDeviceToHost);
//  for(i = 0; i < 100; ++i){
//	  printf("%d  %f  %f\n", i, temp[i], temp1[i]);
//  }


/*
   FILE* fp;
   int conv = 0;
   int connect = 0;
   char weights_path[100];
   float* data;
   for(i = 0; i < net.layer_num; ++i){
 	  struct LayerParameter temp = net.layer_parameter[i];
 	  if(temp.learn_parameter){
 		  if(temp.type == CONVOLUTION)
 			  sprintf(weights_path, "/home/yang/yolo-weights/convolution%d.txt", ++conv);
 		  else
 			  sprintf(weights_path, "/home/yang/yolo-weights/connect%d.txt", ++connect);
 		  printf("%s\n",weights_path);
 		  fp = fopen(weights_path,"r");
 		  data = (float*)malloc(sizeof(float)*net.layer_parameter[i].learn_parameter[1]->count);
 		  fread(data, sizeof(float), net.layer_parameter[i].learn_parameter[1]->count, fp);
 		  cudaMemcpy(net.layer_parameter[i].learn_parameter[1]->gpu_data, data, sizeof(float)*net.layer_parameter[i].learn_parameter[1]->count, cudaMemcpyHostToDevice);
 		  free(data);
 		  data = (float*)malloc(sizeof(float)*net.layer_parameter[i].learn_parameter[0]->count);
 		  fread(data, sizeof(float), net.layer_parameter[i].learn_parameter[0]->count, fp);
 		  cudaMemcpy(net.layer_parameter[i].learn_parameter[0]->gpu_data, data, sizeof(float)*net.layer_parameter[i].learn_parameter[0]->count, cudaMemcpyHostToDevice);
 		  free(data);
 		  fclose(fp);
 	  }
   }
   //加载输入数据
   fp = fopen("/home/yang/Desktop/image_data.txt","r");
   data = (float*)malloc(sizeof(float)*net.inputs[0]->count);
   fread(data, sizeof(float), net.inputs[0]->count, fp);
   cudaMemcpy(net.inputs[0]->gpu_data, data, sizeof(float)*net.inputs[0]->count, cudaMemcpyHostToDevice);
   //cudaMemcpy(net.inputs[0]->gpu_data+net.inputs[0]->count/2, data, sizeof(float)*net.inputs[0]->count/2, cudaMemcpyHostToDevice);
   free(data);
   fclose(fp);
//   start = clock();
//   cudaMalloc(&data, sizeof(float)*500*500*3*21);
//   cudaFree(data);
//   finish = clock();
//   printf( "%f seconds\n",(finish - start) / CLOCKS_PER_SEC);
   printf("start count time\n");
   start = clock();
   forward(net);//这个函数代替下面的注释语句
   finish = clock();
   printf( "%f seconds\n",(finish - start) / CLOCKS_PER_SEC);
   //输出结果对比
   float con_data[1470];
   fp = fopen("/home/yang/Desktop/conv_data.txt","r");
   fread(con_data, sizeof(float), 1470, fp);
   fclose(fp);
   data = (float*)malloc(sizeof(float)*net.outputs[0]->count);
   cudaMemcpy(data, net.outputs[0]->gpu_data, sizeof(float)*net.outputs[0]->count, cudaMemcpyDeviceToHost);
//   for(i = 0; i < 1470; ++i)
//	   //printf("%d %f %f %f %f\n", i , data[i+1470], data[i], con_data[i], data[i]-con_data[i]);
//	   printf("%d %f %f %f\n",i, data[i], con_data[i], data[i]-con_data[i]);
   printf( "%f seconds\n",(finish - start) / CLOCKS_PER_SEC);*/
   deletenet(net);
  //释放内存
  //DeleteLayerName(&names);
  //释放内存结束
   return 1;
}
#endif

#ifdef YOLO
#include <stdio.h>
#include "DL/util/net.h"
#include "DL/util/layer_factory.h"
#include <cuda_runtime.h>
#include <time.h>

void confignet(struct Net* net){
	  net->layer_num = 0;
	  net->tops_num = 0;
	  net->tops = NULL;
	  net->names = NULL;
	  net->layer_parameter = NULL;

	  char* input[1] = {"data"};
	  make_input(net, "data", 1, input, 1, 3, 448, 448);

	  char* b_1[1] = {"data"};
	  char* t_1[1] = {"conv1"};
	  make_convolution(net, "conv1", 1, b_1, 1, t_1, 64, 3, 2, 7, 1);

	  char* b_2[1] = {"conv1"};
	  char* t_2[1] = {"ReLU1"};
	  make_relu(net, "ReLU1", 1, b_2, 1, t_2, 0.1);

	  char* b_3[1] = {"ReLU1"};
	  char* t_3[1] = {"pool1"};
	  make_pooling(net, "pool1", 1, b_3, 1, t_3, 0, 2, 2, MAX, 0);

	  char* b_4[1] = {"pool1"};
	  char* t_4[1] = {"conv2"};
	  make_convolution(net, "conv2", 1, b_4, 1, t_4, 192, 1, 1, 3, 1);

	  char* b_5[1] = {"conv2"};
	  char* t_5[1] = {"ReLU2"};
	  make_relu(net, "ReLU2", 1, b_5, 1, t_5, 0.1);

	  char* b_6[1] = {"ReLU2"};
	  char* t_6[1] = {"pool2"};
	  make_pooling(net, "pool2", 1, b_6, 1, t_6, 0, 2, 2, MAX, 0);

	  char* b_7[1] = {"pool2"};
	  char* t_7[1] = {"conv3"};
	  make_convolution(net, "conv3", 1, b_7, 1, t_7, 128, 0, 1, 1, 1);

	  char* b_8[1] = {"conv3"};
	  char* t_8[1] = {"ReLU3"};
	  make_relu(net, "ReLU3", 1, b_8, 1, t_8, 0.1);

	  char* b_9[1] = {"ReLU3"};
	  char* t_9[1] = {"conv4"};
	  make_convolution(net, "conv4", 1, b_9, 1, t_9, 256, 1, 1, 3, 1);

	  char* b_10[1] = {"conv4"};
	  char* t_10[1] = {"ReLU4"};
	  make_relu(net, "ReLU3", 1, b_10, 1, t_10, 0.1);

	  char* b_11[1] = {"ReLU4"};
	  char* t_11[1] = {"conv5"};
	  make_convolution(net, "conv4", 1, b_11, 1, t_11, 256, 0, 1, 1, 1);

	  char* b_12[1] = {"conv5"};
	  char* t_12[1] = {"ReLU5"};
	  make_relu(net, "ReLU5", 1, b_12, 1, t_12, 0.1);

	  char* b_13[1] = {"ReLU5"};
	  char* t_13[1] = {"conv6"};
	  make_convolution(net, "conv4", 1, b_13, 1, t_13, 512, 1, 1, 3, 1);

	  char* b_14[1] = {"conv6"};
	  char* t_14[1] = {"ReLU6"};
	  make_relu(net, "ReLU6", 1, b_14, 1, t_14, 0.1);

	  char* b_15[1] = {"ReLU6"};
	  char* t_15[1] = {"pool3"};
	  make_pooling(net, "pool3", 1, b_15, 1, t_15, 0, 2, 2, MAX, 0);

	  char* b_16[1] = {"pool3"};
	  char* t_16[1] = {"conv7"};
	  make_convolution(net, "conv7", 1, b_16, 1, t_16, 256, 0, 1, 1, 1);

	  char* b_17[1] = {"conv7"};
	  char* t_17[1] = {"ReLU7"};
	  make_relu(net, "ReLU7", 1, b_17, 1, t_17, 0.1);

	  char* b_18[1] = {"ReLU7"};
	  char* t_18[1] = {"conv8"};
	  make_convolution(net, "conv8", 1, b_18, 1, t_18, 512, 1, 1, 3, 1);

	  char* b_19[1] = {"conv8"};
	  char* t_19[1] = {"ReLU8"};
	  make_relu(net, "ReLU8", 1, b_19, 1, t_19, 0.1);

	  char* b_20[1] = {"ReLU8"};
	  char* t_20[1] = {"conv9"};
	  make_convolution(net, "conv9", 1, b_20, 1, t_20, 256, 0, 1, 1, 1);

	  char* b_21[1] = {"conv9"};
	  char* t_21[1] = {"ReLU9"};
	  make_relu(net, "ReLU9", 1, b_21, 1, t_21, 0.1);

	  char* b_22[1] ={"ReLU9"};
	  char* t_22[1] = {"conv10"};
	  make_convolution(net, "conv10", 1, b_22, 1, t_22, 512, 1, 1, 3, 1);

	  char* b_23[1] ={"conv10"};
	  char* t_23[1] = {"ReLU10"};
	  make_relu(net, "ReLU10", 1, b_23, 1, t_23, 0.1);

	  char* b_24[1] ={"ReLU10"};
	  char* t_24[1] = {"conv11"};
	  make_convolution(net, "conv11", 1, b_24, 1, t_24, 256, 0, 1, 1, 1);

	  char* b_25[1] ={"conv11"};
	  char* t_25[1] = {"ReLU11"};
	  make_relu(net, "ReLU11", 1, b_25, 1, t_25, 0.1);

	  char* b_26[1] ={"ReLU11"};
	  char* t_26[1] = {"conv12"};
	  make_convolution(net, "conv12", 1, b_26, 1, t_26, 512, 1, 1, 3, 1);

	  char* b_27[1] ={"conv12"};
	  char* t_27[1] = {"ReLU12"};
	  make_relu(net, "ReLU12", 1, b_27, 1, t_27, 0.1);

	  char* b_28[1] ={"ReLU12"};
	  char* t_28[1] = {"conv13"};
	  make_convolution(net, "conv13", 1, b_28, 1, t_28, 256, 0, 1, 1, 1);

	  char* b_29[1] ={"conv13"};
	  char* t_29[1] = {"ReLU13"};
	  make_relu(net, "ReLU13", 1, b_29, 1, t_29, 0.1);

	  char* b_30[1] ={"ReLU13"};
	  char* t_30[1] = {"conv14"};
	  make_convolution(net, "conv14", 1, b_30, 1, t_30, 512, 1, 1, 3, 1);

	  char* b_31[1] ={"conv14"};
	  char* t_31[1] = {"ReLU14"};
	  make_relu(net, "ReLU14", 1, b_31, 1, t_31, 0.1);

	  char* b_32[1] ={"ReLU14"};
	  char* t_32[1] = {"conv15"};
	  make_convolution(net, "conv15", 1, b_32, 1, t_32, 512, 0, 1, 1, 1);

	  char* b_33[1] ={"conv15"};
	  char* t_33[1] = {"ReLU15"};
	  make_relu(net, "ReLU15", 1, b_33, 1, t_33, 0.1);

	  char* b_34[1] ={"ReLU15"};
	  char* t_34[1] = {"conv16"};
	  make_convolution(net, "conv16", 1, b_34, 1, t_34, 1024, 1, 1, 3, 1);

	  char* b_35[1] ={"conv16"};
	  char* t_35[1] = {"ReLU16"};
	  make_relu(net, "ReLU16", 1, b_35, 1, t_35, 0.1);

	  char* b_36[1] = {"ReLU16"};
	  char* t_36[1] = {"pool4"};
	  make_pooling(net, "pool4", 1, b_36, 1, t_36, 0, 2, 2, MAX, 0);

	  char* b_37[1] ={"pool4"};
	  char* t_37[1] = {"conv17"};
	  make_convolution(net, "conv17", 1, b_37, 1, t_37, 512, 0, 1, 1, 1);

	  char* b_38[1] ={"conv17"};
	  char* t_38[1] = {"ReLU17"};
	  make_relu(net, "ReLU17", 1, b_38, 1, t_38, 0.1);

	  char* b_39[1] ={"ReLU17"};
	  char* t_39[1] = {"conv18"};
	  make_convolution(net, "conv18", 1, b_39, 1, t_39, 1024, 1, 1, 3, 1);

	  char* b_40[1] ={"conv18"};
	  char* t_40[1] = {"ReLU18"};
	  make_relu(net, "ReLU18", 1, b_40, 1, t_40, 0.1);

	  char* b_41[1] ={"ReLU18"};
	  char* t_41[1] = {"conv19"};
	  make_convolution(net, "conv19", 1, b_41, 1, t_41, 512, 0, 1, 1, 1);

	  char* b_42[1] ={"conv19"};
	  char* t_42[1] = {"ReLU19"};
	  make_relu(net, "ReLU19", 1, b_42, 1, t_42, 0.1);

	  char* b_43[1] ={"ReLU19"};
	  char* t_43[1] = {"conv20"};
	  make_convolution(net, "conv20", 1, b_43, 1, t_43, 1024, 1, 1, 3, 1);

	  char* b_44[1] ={"conv20"};
	  char* t_44[1] = {"ReLU20"};
	  make_relu(net, "ReLU20", 1, b_44, 1, t_44, 0.1);

	  char* b_45[1] ={"ReLU20"};
	  char* t_45[1] = {"conv21"};
	  make_convolution(net, "conv21", 1, b_45, 1, t_45, 1024, 1, 1, 3, 1);

	  char* b_46[1] ={"conv21"};
	  char* t_46[1] = {"ReLU21"};
	  make_relu(net, "ReLU21", 1, b_46, 1, t_46, 0.1);

	  char* b_47[1] ={"ReLU21"};
	  char* t_47[1] = {"conv22"};
	  make_convolution(net, "conv22", 1, b_47, 1, t_47, 1024, 1, 2, 3, 1);

	  char* b_48[1] ={"conv22"};
	  char* t_48[1] = {"ReLU22"};
	  make_relu(net, "ReLU22", 1, b_48, 1, t_48, 0.1);

	  char* b_49[1] ={"ReLU22"};
	  char* t_49[1] = {"conv23"};
	  make_convolution(net, "conv23", 1, b_49, 1, t_49, 1024, 1, 1, 3, 1);

	  char* b_50[1] ={"conv23"};
	  char* t_50[1] = {"ReLU23"};
	  make_relu(net, "ReLU23", 1, b_50, 1, t_50, 0.1);

	  char* b_51[1] ={"ReLU23"};
	  char* t_51[1] = {"conv24"};
	  make_convolution(net, "conv24", 1, b_51, 1, t_51, 1024, 1, 1, 3, 1);

	  char* b_52[1] ={"conv24"};
	  char* t_52[1] = {"ReLU24"};
	  make_relu(net, "ReLU24", 1, b_52, 1, t_52, 0.1);

	  char* b_53[1] = {"ReLU24"};
	  char* t_53[1] = {"connect1"};
	  make_innerproduct(net, "connect1", 1, b_53, 1, t_53, 4096);

	  char* b_54[1] ={"connect1"};
	  char* t_54[1] = {"ReLU25"};
	  make_relu(net, "ReLU25", 1, b_54, 1, t_54, 0.1);

	  char* b_55[1] = {"ReLU25"};
	  char* t_55[1] = {"connect2"};
	  make_innerproduct(net, "connect2", 1, b_55, 1, t_55, 1470);

	  char* inputblobs[1] = {"data"};
	  char* outputblobs[1] = {"ReLU1"};
	  set_net_io(net, 1, inputblobs, 1, outputblobs);
}


int main(){
  int i;
  double start, finish;
  struct Net net;
  cublasStatus_t status;
  status = cublasCreate(&(net.cublas_handle));
  if(status != CUBLAS_STATUS_SUCCESS){
	  printf("get cublas handle error!\n");
  }else{
	  printf("get cublas handle success!\n");
  }
  //struct LayerName names;
  //InitLayerName(&names);
  struct Map_Type_Setup_Forward handle;
  net.handle.num = 0;
  net.handle.functions = NULL;
  regist_layer(&(net.handle));
  confignet(&net);
  setup(net);



   FILE* fp;
   int conv = 0;
   int connect = 0;
   char weights_path[100];
   float* data;
   for(i = 0; i < net.layer_num; ++i){
 	  struct LayerParameter temp = net.layer_parameter[i];
 	  if(temp.learn_parameter){
 		  if(temp.type == CONVOLUTION)
 			  sprintf(weights_path, "/home/yang/yolo-weights/convolution%d.txt", ++conv);
 		  else
 			  sprintf(weights_path, "/home/yang/yolo-weights/connect%d.txt", ++connect);
 		  printf("%s\n",weights_path);
 		  fp = fopen(weights_path,"r");
 		  data = (float*)malloc(sizeof(float)*net.layer_parameter[i].learn_parameter[1]->count);
 		  fread(data, sizeof(float), net.layer_parameter[i].learn_parameter[1]->count, fp);
 		  cudaMemcpy(net.layer_parameter[i].learn_parameter[1]->gpu_data, data, sizeof(float)*net.layer_parameter[i].learn_parameter[1]->count, cudaMemcpyHostToDevice);
 		  free(data);
 		  data = (float*)malloc(sizeof(float)*net.layer_parameter[i].learn_parameter[0]->count);
 		  fread(data, sizeof(float), net.layer_parameter[i].learn_parameter[0]->count, fp);
 		  cudaMemcpy(net.layer_parameter[i].learn_parameter[0]->gpu_data, data, sizeof(float)*net.layer_parameter[i].learn_parameter[0]->count, cudaMemcpyHostToDevice);
 		  free(data);
 		  fclose(fp);
 	  }
   }
   //加载输入数据
   fp = fopen("/home/yang/Desktop/image_data.txt","r");
   data = (float*)malloc(sizeof(float)*net.inputs[0]->count);
   fread(data, sizeof(float), net.inputs[0]->count, fp);
   cudaMemcpy(net.inputs[0]->gpu_data, data, sizeof(float)*net.inputs[0]->count, cudaMemcpyHostToDevice);
   //cudaMemcpy(net.inputs[0]->gpu_data+net.inputs[0]->count/2, data, sizeof(float)*net.inputs[0]->count/2, cudaMemcpyHostToDevice);
   free(data);
   fclose(fp);
//   start = clock();
//   cudaMalloc(&data, sizeof(float)*500*500*3*21);
//   cudaFree(data);
//   finish = clock();
//   printf( "%f seconds\n",(finish - start) / CLOCKS_PER_SEC);
   printf("start count time\n");
   start = clock();
   forward(net);//这个函数代替下面的注释语句
   finish = clock();
   printf( "%f seconds\n",(finish - start) / CLOCKS_PER_SEC);
   //输出结果对比
   float con_data[1470];
   fp = fopen("/home/yang/Desktop/conv_data.txt","r");
   fread(con_data, sizeof(float), 1470, fp);
   fclose(fp);
   data = (float*)malloc(sizeof(float)*net.outputs[0]->count);
   cudaMemcpy(data, net.outputs[0]->gpu_data, sizeof(float)*net.outputs[0]->count, cudaMemcpyDeviceToHost);
   for(i = 0; i < 1470; ++i)
	   //printf("%d %f %f %f %f\n", i , data[i+1470], data[i], con_data[i], data[i]-con_data[i]);
	   printf("%d %f %f %f\n",i, data[i], con_data[i], data[i]-con_data[i]);
   printf( "%f seconds\n",(finish - start) / CLOCKS_PER_SEC);
   deletenet(net);
  //释放内存
  //DeleteLayerName(&names);
  //释放内存结束
   return 1;
}

#endif


