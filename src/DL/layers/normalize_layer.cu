#include "hip/hip_runtime.h"
#include "DL/layers/normalize_layer.h"
#include "DL/util/common_function.h"
#include <cblas.h>
#include <malloc.h>
#include <stdlib.h>

void setup_normalize(struct LayerParameter* layer_parameter){
	struct Blob* bottom = layer_parameter->bottoms[0];
	int n = bottom->n;
	int c = bottom->c;
	int h = bottom->h;
	int w = bottom->w;
	struct Blob* learn_param = layer_parameter->learn_parameter[0];
	MakeBlob(n, c, h, w, layer_parameter->tops[0]);
	if(layer_parameter->parameter.normalize_param.channel_shared){
		MakeBlob(1, 1, 1, 1, learn_param);
	}else{
		MakeBlob(1, c, 1, 1, learn_param);
	}
	Doutput_shape_info();
}

__global__ void normalize_across_spatial(float* input, int n, int dim){
	int idx = threadIdx.x;
	float sum = 1e-10;
	int i = 0;
	if(idx < n){
		int temp = n*dim;
		for(i = 0; i < dim; ++i){
			sum+=input[temp+i];
		}
		//sum = pow(sum, float(0.5));
		for(i = 0; i < dim; ++i){
			input[temp+i]/=sum;
			input[temp+i]=pow(input[temp+i], float(0.5));
		}
	}
}
//
//__global__ void normalize_nacross_spatial(float* input, int n, int c, int dim){
//	if(blockIdx.x < n && threadIdx.x < dim){
//		float sum = 1e-10;
//		int i = 0;
//		int temp = dim*blockIdx.x*c+threadIdx.x;
//		for(i = 0; i < c; ++i){
//			sum = sum + input[temp+i*dim];
//		}
//		for(i = 0; i < c; ++i){
//			input[temp+i*dim] = input[temp+i*dim] / sum;
//			input[temp+i*dim]=pow(input[temp+i*dim], float(0.5));
//		}
//	}
//}



__global__ void normalize_nacross_spatial(float* input, int n, int c, int h, int w ){
	if(blockIdx.x < h && threadIdx.x < w){
		float sum = 1e-10;
		int i = 0;
		//int temp = dim*(blockIdx.x*blockDim.x + threadIdx.x);
		int temp = blockIdx.x*blockDim.x+threadIdx.x;
		for(i = 0; i < c; ++i){
			sum= sum + input[temp+i*h*w];
		}
		for(i = 0; i < c; ++i){
			input[temp+i*h*w]= input[temp+i*h*w] / sum;
			input[temp+i*h*w]=pow(input[temp+i*h*w], float(0.5));
		}
	}
}


__global__ void normalize_channel_nshared(float* input, int n, int c, int dim, float* scale){
	if(blockIdx.x < n && threadIdx.x < c)
	{
		int i = 0;
		int temp = dim*(blockIdx.x*blockDim.x + threadIdx.x);
		for(i = 0; i < dim; ++i){
			input[temp+i] = input[temp+i] * scale[threadIdx.x];
		}
	}
}

void forward_normalize(struct LayerParameter* layer_parameter){
	Doutput_info();
	struct Blob* input = layer_parameter->bottoms[0];
	struct Blob* output = layer_parameter->tops[0];
	struct Blob* scale = layer_parameter->learn_parameter[0];
	int n,c,h,w;
	n = input->n;
	c = input->c;
	h = input->h;
	w = input->w;
	power_gpu(input->gpu_data, output->gpu_data, input->count);
//	float* data_input = (float*)malloc(sizeof(float)*input->count);
//	float* data_output = (float*)malloc(sizeof(float)*input->count);
//	hipMemcpy(data_input, input->gpu_data, sizeof(float)*input->count, hipMemcpyDeviceToHost);
//	hipMemcpy(data_output, output->gpu_data, sizeof(float)*input->count, hipMemcpyDeviceToHost);
//	int j = 0;
//	for(; j < input->count; ++j){
//		printf("%f %f %f\n", data_input[j], data_output[j], data_input[j]*data_input[j]-data_output[j]);
//	}
	int across_spatial = layer_parameter->parameter.normalize_param.across_spatial;
	int channel_shared = layer_parameter->parameter.normalize_param.channel_shared;
	if(across_spatial){
		normalize_across_spatial<<<1, n>>>(output->gpu_data, n, c*h*w);
	}else{
		//dim3 block((h*w+CUDA_NUM_THREADS-1)/CUDA_NUM_THREADS,CUDA_NUM_THREADS);
		//normalize_nacross_spatial<<<n, block>>>(output->gpu_data, n, c, h*w);
//		float* sumt;
//		hipMalloc(&sumt, sizeof(float)*n*h*w);
		int i = 0;
		for(i = 0; i < n; ++i){
			normalize_nacross_spatial<<<h, w>>>(output->gpu_data+i*n*c*h*w, n, c, h, w);
		}
//		float* data = (float*)malloc(sizeof(float)*n*h*w);;
//		hipMemcpy(data, sumt, sizeof(float)*n*h*w, hipMemcpyDeviceToHost);
//		int j;
//		for(j = 0; j < n*h*w; ++j){
//			printf("%d %f\n", j, data[j]);
//		}
	}
	if(channel_shared){
		scale_gpu(output->gpu_data, output->gpu_data, output->count, scale->gpu_data[0]);
	}else{
		normalize_channel_nshared<<<n, c>>>(output->gpu_data, n, c, h*w, scale->gpu_data);
		float* data = (float*)malloc(sizeof(float)*scale->count);
		hipMemcpy(data, scale->gpu_data, sizeof(float)*scale->count, hipMemcpyDeviceToHost);
//		int j = 0;
//		for(j = 0; j < scale->count; ++j){
//			printf("sacle %d %f\n", j, data[j]);
//		}
		free(data);
	}

//	struct Blob* bottom = layer_parameter->bottoms[0];
//	int n = bottom->n;
//	int c = bottom->c;
//	int h = bottom->h;
//	int w = bottom->w;
//	int dim = c*h*w;
//	int spatial_dim = h*w;
//	int i, j;
//	float* bottom_data = bottom->cpu_data;
//	float* top_data = layer_parameter->tops[0]->cpu_data;
//	float* med = (float*)malloc(sizeof(float)*dim);
//	float* norm;
//	float* norm_ori;
//	float* sum_multiplier = (float*)malloc(sizeof(float)*c);
//	float* sum_spatial_multiplier = (float*)malloc(sizeof(float)*spatial_dim);
//	float* scale = layer_parameter->learn_parameter[0]->cpu_data;
//	int across_spatial = layer_parameter->parameter.normalize_param.across_spatial;
//	int channel_shared = layer_parameter->parameter.normalize_param.channel_shared;
//	for(i = 0; i < c; ++i)
//		sum_multiplier[i] = 1.0;
//	for(i = 0; i < spatial_dim; ++i)
//		sum_spatial_multiplier[i] = 1.0;
//	if(across_spatial){
//		norm = (float*)malloc(sizeof(float)*n);
//		for(i = 0; i < n; ++i)
//			norm[i] = 1e-10;
//	}else{
//		norm = (float*)malloc(sizeof(float)*n*h*w);
//		for(i = n*h*w-1; i >= 0; --i)
//			norm[i] = 1e-10;
//	}
//	norm_ori = norm;
//	for(i = 0; i < n; ++i){
//		for(j = 0; j < dim; ++j){
//			med[j] = bottom_data[j]*bottom_data[j];
//		}
//		if(across_spatial){
//			for(j = 0; j < dim; ++j)
//				norm[i] += med[j];
//			norm[i] = pow(norm[i], 0.5);
//			for(j = 0; j < dim; ++j){
//				top_data[j] = bottom_data[j]/norm[i];
//			}
//		}else{
//			cblas_sgemm(CblasRowMajor, CblasNoTrans, CblasNoTrans, 1, spatial_dim, c,
//					1.0, sum_multiplier, c, med, spatial_dim, 0.0, norm, spatial_dim);
//
//			for(j = 0; j < spatial_dim; ++j)
//				norm[j] = pow(norm[j], 0.5);
//			cblas_sgemm(CblasRowMajor, CblasNoTrans, CblasNoTrans, c, spatial_dim, 1,
//					1.0, sum_multiplier, 1, norm, spatial_dim, 0.0, med, spatial_dim);
//			for(j = 0; j < dim; ++j){
//				top_data[j] = bottom_data[j]/med[j];
//			}
//			norm = norm + spatial_dim;
//		}
//		if(channel_shared){
//			for(j = 0; j < dim; ++j)
//				top_data[j] = top_data[j]*scale[0];
//		}else{
//			cblas_sgemm(CblasRowMajor, CblasNoTrans, CblasNoTrans, c, spatial_dim, 1,
//					1.0, scale, 1, sum_spatial_multiplier, spatial_dim, 0.0, med, spatial_dim);
//			for(j = 0; j < dim; ++j){
//				top_data[j] = top_data[j]*med[j];
//			}
//		}
//		bottom_data += dim;
//		top_data += dim;
//	}
//	free(med);
//	free(norm_ori);
//	free(sum_multiplier);
//	free(sum_spatial_multiplier);
}
