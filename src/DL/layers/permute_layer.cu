#include "hip/hip_runtime.h"
#include "DL/layers/permute_layer.h"
#include "DL/util/common_function.h"

void setup_permute(struct LayerParameter* layer_parameter){
	int idx_n = layer_parameter->parameter.permute_param.idx_n;
	int idx_c = layer_parameter->parameter.permute_param.idx_c;
	int idx_h = layer_parameter->parameter.permute_param.idx_h;
	int idx_w = layer_parameter->parameter.permute_param.idx_w;
	int shape[4] = {layer_parameter->bottoms[0]->n, layer_parameter->bottoms[0]->c, layer_parameter->bottoms[0]->h, layer_parameter->bottoms[0]->w};
	int n = shape[idx_n];
	int c = shape[idx_c];
	int h = shape[idx_h];
	int w = shape[idx_w];
	MakeBlob(n, c, h, w, layer_parameter->tops[0]);
	Doutput_shape_info();
}

__global__ void permute_kernel(float* input, int in_n, int in_c, int in_h, int in_w,
		float* output, int out_n, int out_c, int out_h, int out_w,
		int idx_n, int idx_c, int idx_h, int idx_w){
	//int idx[4] = { blockIdx.y, blockIdx.x, threadIdx.x, threadIdx.y};
	int index = blockIdx.x*blockDim.x+threadIdx.x;
	int i_n, i_c, i_h, i_w;
	int temp = in_c*in_h*in_w;
	i_n = index/temp;
	temp = index%temp;
	i_c = temp/(in_w*in_h);
	temp = temp%(in_w*in_h);
	i_h = temp/in_w;
	i_w = temp%in_w;
	//int idx[4] = {blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y};
	int idx[4] = {i_n, i_c, i_h, i_w};
	if(idx[0] < in_n && idx[1] < in_c && idx[2] < in_h && idx[3] < in_w){

		int idx_input = ((idx[0]*in_c+idx[1])*in_h+idx[2])*in_w+idx[3];

		int idx_output = ((idx[idx_n]*out_c+idx[idx_c])*out_h+idx[idx_h])*out_w+idx[idx_w];
		output[idx_output] = input[idx_input];
	}
}

void forward_permute(struct LayerParameter* layer_parameter){
	Doutput_info();
//	int idx_n = layer_parameter->parameter.permute_param.idx_n;
//	int idx_c = layer_parameter->parameter.permute_param.idx_c;
//	int idx_h = layer_parameter->parameter.permute_param.idx_h;
//	int idx_w = layer_parameter->parameter.permute_param.idx_w;
//	int shape_bottom[4] = {layer_parameter->bottoms[0]->n, layer_parameter->bottoms[0]->c,
//			layer_parameter->bottoms[0]->h, layer_parameter->bottoms[0]->w};
//	int shape_top[4] = {shape_bottom[idx_n], shape_bottom[idx_c], shape_bottom[idx_h], shape_bottom[idx_w]};
//	float* top_data = layer_parameter->tops[0]->cpu_data;
//	float* bottom_data = layer_parameter->bottoms[0]->cpu_data;
//	int n,c,h,w;
//	int top_idx,bottom_idx;
//	int b_flex[4];
//	int f_flex[4] = {idx_n, idx_c, idx_h, idx_w};
//	for(n = 0; n < shape_top[0]; ++n){
//		for(c = 0; c < shape_top[1]; ++c){
//			for(h = 0; h < shape_top[2]; ++h){
//				for(w = 0; w < shape_top[3]; ++w){
//					b_flex[f_flex[0]] = n;
//					b_flex[f_flex[1]] = c;
//					b_flex[f_flex[2]] = h;
//					b_flex[f_flex[3]] = w;
//					top_idx = ((n*shape_top[1]+c)*shape_top[2]+h)*shape_top[3]+w;
//					bottom_idx = ((b_flex[0]*shape_bottom[1]+b_flex[1])*shape_bottom[2]+b_flex[2])*shape_bottom[3]+b_flex[3];
//					top_data[top_idx] = bottom_data[bottom_idx];
//				}
//			}
//		}
//	}
	int in_n, in_c, in_h, in_w;
	int out_n, out_c, out_h, out_w;
	in_n = layer_parameter->bottoms[0]->n;
	in_c = layer_parameter->bottoms[0]->c;
	in_h = layer_parameter->bottoms[0]->h;
	in_w = layer_parameter->bottoms[0]->w;
	out_n = layer_parameter->tops[0]->n;
	out_c = layer_parameter->tops[0]->c;
	out_h = layer_parameter->tops[0]->h;
	out_w = layer_parameter->tops[0]->w;
	//dim3 grid((in_n+CUDA_NUM_THREADS-1)/CUDA_NUM_THREADS, (in_c+CUDA_NUM_THREADS-1)/CUDA_NUM_THREADS);
	//dim3 block((in_h+CUDA_NUM_THREADS-1)/CUDA_NUM_THREADS, (in_w+CUDA_NUM_THREADS-1)/CUDA_NUM_THREADS);
//	dim3 grid(2147483647, 65535, 65535);
//	dim3 block(10224);
//	printf("permute dimension %d %d %d %d\n", in_n, in_c, in_h, in_w);
//	printf("permute dimension %d %d %d %d\n", layer_parameter->parameter.permute_param.idx_n, layer_parameter->parameter.permute_param.idx_c,
//			layer_parameter->parameter.permute_param.idx_h, layer_parameter->parameter.permute_param.idx_w);
	//for(;;);
	//copy(layer_parameter->bottoms[0]->gpu_data, layer_parameter->tops[0]->gpu_data, layer_parameter->tops[0]->count);
	permute_kernel<<<(layer_parameter->bottoms[0]->count+CUDA_NUM_THREADS-1)/CUDA_NUM_THREADS, CUDA_NUM_THREADS>>>(layer_parameter->bottoms[0]->gpu_data, in_n, in_c, in_h, in_w,
			layer_parameter->tops[0]->gpu_data, out_n, out_c, out_h, out_w,
			layer_parameter->parameter.permute_param.idx_n, layer_parameter->parameter.permute_param.idx_c,
			layer_parameter->parameter.permute_param.idx_h, layer_parameter->parameter.permute_param.idx_w);
//	permute_kernel<<<1, block>>>(layer_parameter->bottoms[0]->gpu_data, in_n, in_c, in_h, in_w,
//				layer_parameter->tops[0]->gpu_data, out_n, out_c, out_h, out_w,
//				layer_parameter->parameter.permute_param.idx_n, layer_parameter->parameter.permute_param.idx_c,
//				layer_parameter->parameter.permute_param.idx_h, layer_parameter->parameter.permute_param.idx_w);
//	hipError_t cudaerror = hipPeekAtLastError();
//	if(cudaerror == hipSuccess){
//		printf("success\n");
//	}else{
//		printf("%s \n", hipGetErrorString(cudaerror));
//	}
//	for(;;);
}
