#include "hip/hip_runtime.h"
#include "DL/layers/pooling_layer.h"
#include <float.h>

void setup_pooling(struct LayerParameter* layer_parameter){
	layer_parameter->has_learn_parameter = 0;
	if(layer_parameter->parameter.pooling_param.global_pooling){
		layer_parameter->parameter.pooling_param.kernel_size = layer_parameter->bottoms[0]->h;
	}
	struct PoolingParameter pooling_param = layer_parameter->parameter.pooling_param;
	struct Blob* input = layer_parameter->bottoms[0];
	struct Blob* output = layer_parameter->tops[0];
	output->n = input->n;
	output->c = input->c;
	output->w = ceil((0.0 + input->w + pooling_param.pad*2 - pooling_param.kernel_size)/pooling_param.stride)+1;
	output->h = ceil((0.0 + input->h + pooling_param.pad*2 - pooling_param.kernel_size)/pooling_param.stride)+1;
	MakeBlob(output->n, output->c, output->h, output->w, output);
	Doutput_shape_info();
}

__global__ void pooling_max_kernel(float* input, int kernel_size, int in_n, int in_c, int in_h, int in_w, int pad, int stride,
		float* output, int out_h, int out_w, int count){
	int idx_idx = blockIdx.x*blockDim.x+threadIdx.x;
	if(idx_idx < count){
		int n,c,h,w;
		int temp1 = (in_c*out_h* out_w);
		int temp2;
		n = idx_idx/temp1;
		temp2 = idx_idx%temp1;
		temp1 = out_h* out_w;
		c = temp2/temp1;
		temp2 = temp2%temp1;
		h = temp2/out_h;
		w = temp2%out_h;
		int speed_up = n*in_c+c;
		int h_ori_temp = h*stride-pad;
		int w_ori_temp = w*stride-pad;
		float max = -FLT_MAX;
		float med;
		int rh, rw;
		int h_ori, w_ori;
		for(rh = 0; rh < kernel_size; ++rh){
			h_ori = h_ori_temp+rh;
			if (h_ori < 0 || h_ori > in_h-1) continue;
			for(rw = 0; rw < kernel_size; ++rw){
				w_ori = w_ori_temp+rw;
				if (w_ori < 0 || w_ori > in_w-1) continue;
				med = input[(speed_up*in_h+h_ori)*in_w+w_ori];
				if (max < med){
					max = med;
				}
			}
		}
		output[idx_idx] = max;
	}
}

__global__ void pooling_max_kernel_new(float* input, int kernel_size, int in_n, int in_c, int in_h, int in_w, int pad, int stride,
		float* output, int out_h, int out_w, int count){
	int n,c,h,w;
	n = blockIdx.x;
	c = blockIdx.y;
	h = threadIdx.x;
	w = threadIdx.y;
	int idx_idx = ((n*in_c+c)*out_h+h)*out_w+w;
	if(n < in_c && c < in_c && h < out_h && w < out_w){
		int speed_up = n*in_c+c;
		int h_ori_temp = h*stride-pad;
		int w_ori_temp = w*stride-pad;
		float max = -FLT_MAX;
		float med;
		int rh, rw;
		int h_ori, w_ori;
		for(rh = 0; rh < kernel_size; ++rh){
			h_ori = h_ori_temp+rh;
			if (h_ori < 0 || h_ori > in_h-1) continue;
			for(rw = 0; rw < kernel_size; ++rw){
				w_ori = w_ori_temp+rw;
				if (w_ori < 0 || w_ori > in_w-1) continue;
				med = input[(speed_up*in_h+h_ori)*in_w+w_ori];
				if (max < med){
					max = med;
				}
			}
		}
		output[idx_idx] = max;
	}
}

__global__ void pooling_ave_kernel_new(float* input, int kernel_size, int in_n, int in_c, int in_h, int in_w, int pad, int stride,
		float* output, int out_h, int out_w, int count){
		int n,c,h,w;
		n = blockIdx.x;
		c = blockIdx.y;
		h = threadIdx.x;
		w = threadIdx.y;
		int idx_idx = ((n*in_c+c)*out_h+h)*out_w+w;
		if(n < in_c && c < in_c && h < out_h && w < out_w){
				int speed_up = n*in_c+c;
				int h_ori_temp = h*stride-pad;
				int w_ori_temp = w*stride-pad;
				int rh, rw;
				int h_ori, w_ori;
				float sum = 0.0;
				for(rh = 0; rh < kernel_size; ++rh){
					h_ori = h_ori_temp+rh;
					if (h_ori < 0 || h_ori > in_h-1) continue;
					for(rw = 0; rw < kernel_size; ++rw){
						w_ori = w_ori_temp+rw;
						if (w_ori < 0 || w_ori > in_w-1) continue;
						sum += input[(speed_up*in_h+h_ori)*in_w+w_ori];
					}
				}
				output[idx_idx] = sum/(kernel_size*kernel_size);
			}
}

__global__ void pooling_ave_kernel(float* input, int kernel_size, int in_n, int in_c, int in_h, int in_w, int pad, int stride,
		float* output, int out_h, int out_w, int count){
	int idx_idx = blockIdx.x*blockDim.x+threadIdx.x;
	if(idx_idx < count){
		int n,c,h,w;
		int temp1 = (in_c*out_h* out_w);
		int temp2;
		n = idx_idx/temp1;
		temp2 = idx_idx%temp1;
		temp1 = out_h* out_w;
		c = temp2/temp1;
		temp2 = temp2%temp1;
		h = temp2/out_h;
		w = temp2%out_h;

		int speed_up = n*in_c+c;
		int h_ori_temp = h*stride-pad;
		int w_ori_temp = w*stride-pad;
		float med;
		int rh, rw;
		int h_ori, w_ori;
		float sum = 0.0;
		for(rh = 0; rh < kernel_size; ++rh){
			h_ori = h_ori_temp+rh;
			if (h_ori < 0 || h_ori > in_h-1) continue;
			for(rw = 0; rw < kernel_size; ++rw){
				w_ori = w_ori_temp+rw;
				if (w_ori < 0 || w_ori > in_w-1) continue;
				med = input[(speed_up*in_h+h_ori)*in_w+w_ori];
				sum += med;
			}
		}
		output[idx_idx] = sum/(kernel_size*kernel_size);
	}
}

void forward_pooling(struct LayerParameter* layer_parameter){
	Doutput_info();
	struct Blob* input = layer_parameter->bottoms[0];
	struct Blob* output = layer_parameter->tops[0];
	struct PoolingParameter pooling_param = layer_parameter->parameter.pooling_param;
	int pad = pooling_param.pad;
	int stride = pooling_param.stride;
	int kernel_size = pooling_param.kernel_size;
	int out_h = output->h;
	int out_w = output->w;
	int in_h = input->h;
	int in_w = input->w;
	int in_c = input->c;
	int in_n = input->n;
	if(layer_parameter->parameter.pooling_param.pooling_type == MAX){
		pooling_max_kernel<<<(output->count+CUDA_NUM_THREADS-1)/CUDA_NUM_THREADS, CUDA_NUM_THREADS>>>(input->gpu_data, kernel_size, in_n, in_c, in_h, in_w, pad, stride,
				output->gpu_data, out_h, out_w, output->count);
//		dim3  grid((in_n+CUDA_NUM_THREADS-1)/CUDA_NUM_THREADS,(in_c+CUDA_NUM_THREADS-1)/CUDA_NUM_THREADS);
//		dim3 block((out_h+CUDA_NUM_THREADS-1)/CUDA_NUM_THREADS,(out_w+CUDA_NUM_THREADS-1)/CUDA_NUM_THREADS);
//		pooling_max_kernel_new<<<grid, block>>>(input->gpu_data, kernel_size, in_n, in_c, in_h, in_w, pad, stride,
//						output->gpu_data, out_h, out_w, output->count);
	}else{
		if(layer_parameter->parameter.pooling_param.pooling_type == AVE){
			pooling_ave_kernel<<<(output->count+CUDA_NUM_THREADS-1)/CUDA_NUM_THREADS, CUDA_NUM_THREADS>>>(input->gpu_data, kernel_size, in_n, in_c, in_h, in_w, pad, stride,
					output->gpu_data, out_h, out_w, output->count);
//			dim3  grid((in_n+CUDA_NUM_THREADS-1)/CUDA_NUM_THREADS,(in_c+CUDA_NUM_THREADS-1)/CUDA_NUM_THREADS);
//			dim3 block((out_h+CUDA_NUM_THREADS-1)/CUDA_NUM_THREADS,(out_w+CUDA_NUM_THREADS-1)/CUDA_NUM_THREADS);
//			pooling_ave_kernel_new<<<grid, block>>>(input->gpu_data, kernel_size, in_n, in_c, in_h, in_w, pad, stride,
//									output->gpu_data, out_h, out_w, output->count);
		}
	}
}
