#include "hip/hip_runtime.h"
#include "DL/layers/priorbox_layer.h"
#include <math.h>
void setup_priorbox(struct LayerParameter* layer_parameter){
	//bottom[0]表示输入的featuremap， bottom[1]表示输入的图像
	int h = layer_parameter->bottoms[0]->h;
	int w = layer_parameter->bottoms[0]->w;
	int shape[4] = {1, 2, h*w*4*layer_parameter->parameter.priorbox_param.priorbox_num, 1};
	MakeBlob(shape[0], shape[1], shape[2], shape[3], layer_parameter->tops[0]);
	Doutput_shape_info();
}

//__global__ void priorbox_kernel(float* top_data, int min_size_, int max_size_, int layer_width, int layer_height,
//		int img_width, int img_height, int step_x, int step_y, int dim, int aspect_num, int num_prior,
//		float* v, float* aspect){
//	int id = blockIdx.x*blockDim.x+threadIdx.x;
//	int h = id/layer_width;
//	int w = id%layer_width;
//	int i,j;
//	if(h < layer_height && w < layer_width){
//		int idx = id*num_prior*4;
//		float center_x,center_y, box_width, box_height;
//		center_x = (w + 0.5) * step_x;
//		center_y = (h + 0.5) * step_y;
//		box_width = box_height = min_size_;
//		top_data[idx++] = (center_x - box_width / 2.) / img_width;
//		top_data[idx++] = (center_y - box_height / 2.) / img_height;
//		top_data[idx++] = (center_x + box_width / 2.) / img_width;
//		top_data[idx++] = (center_y + box_height / 2.) / img_height;
//		if(max_size_ != -1){
//			box_width = box_height = sqrt(float(min_size_ * max_size_));
//			top_data[idx++] = (center_x - box_width / 2.) / img_width;
//			top_data[idx++] = (center_y - box_height / 2.) / img_height;
//			top_data[idx++] = (center_x + box_width / 2.) / img_width;
//			top_data[idx++] = (center_y + box_height / 2.) / img_height;
//		}
//		for(i = 0; i < aspect_num; ++i){
//			box_width = min_size_ * sqrt(aspect[i]);
//			box_height = min_size_ / sqrt(aspect[i]);
//			top_data[idx++] = (center_x - box_width / 2.) / img_width;
//			top_data[idx++] = (center_y - box_height / 2.) / img_height;
//			top_data[idx++] = (center_x + box_width / 2.) / img_width;
//			top_data[idx++] = (center_y + box_height / 2.) / img_height;
//		}
//		idx = id*num_prior*4+dim;
//		for(i = 0; i < num_prior; ++i){
//			for(j = 0; j < 4; ++j){
//				top_data[idx++] = v[j];
//			}
//		}
//	}
//}

void forward_priorbox(struct LayerParameter* layer_parameter){
//	Doutput_info();
//	int min_size_ = layer_parameter->parameter.priorbox_param.min_size;
//	int max_size_ = layer_parameter->parameter.priorbox_param.max_size;
//	int layer_width = layer_parameter->bottoms[0]->w;
//	int layer_height = layer_parameter->bottoms[0]->h;
//	int img_width = layer_parameter->bottoms[1]->w;
//	int img_height = layer_parameter->bottoms[1]->h;
//	float step_x = 1.0*img_width/layer_width;
//	float step_y = 1.0*img_height/layer_height;
//	float* top_data = layer_parameter->tops[0]->gpu_data;
//	int dim = layer_height*layer_width*4*layer_parameter->parameter.priorbox_param.priorbox_num;
////	int idx = 0;
////	int h, w, i, j;
//	int aspect_num = layer_parameter->parameter.priorbox_param.aspect_num;
//	int num_prior = layer_parameter->parameter.priorbox_param.priorbox_num;
//	float* v = layer_parameter->parameter.priorbox_param.variance;
//	float* aspect = layer_parameter->parameter.priorbox_param.aspect;
//	priorbox_kernel<<<(layer_width*layer_height+CUDA_NUM_THREADS-1)/CUDA_NUM_THREADS, CUDA_NUM_THREADS>>>(top_data,min_size_, max_size_, layer_width, layer_height,
//			img_width, img_height, step_x, step_y, dim, aspect_num, num_prior,
//			v, aspect);
//	hipError_t cudaerror = hipPeekAtLastError();
//	if(cudaerror == hipSuccess){
//		printf("success\n");
//	}else{
//		printf("%s \n", hipGetErrorString(cudaerror));
//	}
//	for(;;);

	Doutput_info();
	int min_size_ = layer_parameter->parameter.priorbox_param.min_size;
	int max_size_ = layer_parameter->parameter.priorbox_param.max_size;
	int layer_width = layer_parameter->bottoms[0]->w;
	int layer_height = layer_parameter->bottoms[0]->h;
	int img_width = layer_parameter->bottoms[1]->w;
	int img_height = layer_parameter->bottoms[1]->h;
	float step_x = 1.0*img_width/layer_width;
	float step_y = 1.0*img_height/layer_height;
	float* top_data = (float*)malloc(sizeof(float)*layer_parameter->tops[0]->count);//layer_parameter->tops[0]->cpu_data;
	float* top_data_ori = top_data;
	int dim = layer_height*layer_width*4*layer_parameter->parameter.priorbox_param.priorbox_num;
	int idx = 0;
	int h, w, i, j;
	int aspect_num = layer_parameter->parameter.priorbox_param.aspect_num;
	int num_prior = layer_parameter->parameter.priorbox_param.priorbox_num;
	float* v = layer_parameter->parameter.priorbox_param.variance;
	float center_x,center_y, box_width, box_height;
	for(h = 0; h < layer_height; ++h){
		for(w = 0; w < layer_width; ++w){
			center_x = (w + 0.5) * step_x;
			center_y = (h + 0.5) * step_y;
			box_width = box_height = min_size_;
			top_data[idx++] = (center_x - box_width / 2.) / img_width;
			top_data[idx++] = (center_y - box_height / 2.) / img_height;
			top_data[idx++] = (center_x + box_width / 2.) / img_width;
			top_data[idx++] = (center_y + box_height / 2.) / img_height;
			if(max_size_ != -1){
				box_width = box_height = sqrt(min_size_ * max_size_);
				top_data[idx++] = (center_x - box_width / 2.) / img_width;
				top_data[idx++] = (center_y - box_height / 2.) / img_height;
				top_data[idx++] = (center_x + box_width / 2.) / img_width;
				top_data[idx++] = (center_y + box_height / 2.) / img_height;
			}
			for(i = 0; i < aspect_num; ++i){
				box_width = min_size_ * sqrt(layer_parameter->parameter.priorbox_param.aspect[i]);
				box_height = min_size_ / sqrt(layer_parameter->parameter.priorbox_param.aspect[i]);
				top_data[idx++] = (center_x - box_width / 2.) / img_width;
				top_data[idx++] = (center_y - box_height / 2.) / img_height;
				top_data[idx++] = (center_x + box_width / 2.) / img_width;
				top_data[idx++] = (center_y + box_height / 2.) / img_height;
			}
		}
	}
	for(i = 0; i < dim; ++i){
		if(top_data[i] > 1.0)
			top_data[i] = 1.0;
		else
			if(top_data[i] < 0.0)
				top_data[i] = 0.0;
	}
	top_data += dim;
	idx = 0;
	for(h = 0; h < layer_height; ++h){
		for(w = 0; w < layer_width; ++w){
			for(i = 0; i < num_prior; ++i){
				for(j = 0; j < 4; ++j){
					top_data[idx++] = v[j];
				}
			}
		}
	}
	hipMemcpy(layer_parameter->tops[0]->gpu_data, top_data_ori, sizeof(float)*layer_parameter->tops[0]->count, hipMemcpyHostToDevice);
	free(top_data_ori);
}
