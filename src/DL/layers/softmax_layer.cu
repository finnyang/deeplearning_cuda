#include "hip/hip_runtime.h"
#include "DL/layers/softmax_layer.h"
#include <cblas.h>
#include <malloc.h>
#include <math.h>
#include "DL/util/common_function.h"

void setup_softmax(struct LayerParameter* layer_parameter){
	MakeBlob(layer_parameter->bottoms[0]->n, layer_parameter->bottoms[0]->c, layer_parameter->bottoms[0]->h,
			layer_parameter->bottoms[0]->w, layer_parameter->tops[0]);
	int outer_num, inter_num, i;
	int axis = layer_parameter->parameter.softmax_param.axis;
	int shape[4] = {layer_parameter->bottoms[0]->n, layer_parameter->bottoms[0]->c, layer_parameter->bottoms[0]->h,
				layer_parameter->bottoms[0]->w};
	outer_num = 1;
	inter_num = 1;
	for(i = 0; i < axis; ++i)
		outer_num *= shape[i];
	for(i = axis+1; i < 4; ++i)
		inter_num *= shape[i];
	MakeBlob(inter_num, outer_num, 1, 1, layer_parameter->meds[0]);
	Doutput_shape_info();
}

__global__ void kernel_max(float* input, int inter_num, int outer_num, int channels, float* max_){
	int id = blockIdx.x*blockDim.x+threadIdx.x;
	int inter_idx = id%inter_num;
	int outer_idx = id/inter_num;
	if(inter_idx < inter_num && outer_idx < outer_num) {
		int idx;
		int i;
		int temp = channels*outer_idx*inter_num;
		idx = outer_idx*inter_num+inter_idx;
		max_[idx] = input[temp];
		for(i = 1; i < channels; ++i){
			if(max_[idx] < input[temp+i]){
				max_[idx] = input[temp+i];
			}
		}
	}
}

__global__ void kernel_minus_max(float* input ,int inter_num, int outer_num, int channels, float* max_){
	int id = blockIdx.x*blockDim.x+threadIdx.x;
	int inter_idx = id%inter_num;
	int outer_idx = id/inter_num;
	if(inter_idx < inter_num && outer_idx < outer_num) {
		int idx;
		int i;
		int temp = channels*outer_idx*inter_num;
		idx = outer_idx*inter_num+inter_idx;
		for(i = 0; i < channels; ++i){
			input[temp+i]-= max_[idx];
		}
	}
}

__global__ void kernel_sum(float* input ,int inter_num, int outer_num, int channels, float* sum_){
	int id = blockIdx.x*blockDim.x+threadIdx.x;
	int inter_idx = id%inter_num;
	int outer_idx = id/inter_num;
	if(inter_idx < inter_num && outer_idx < outer_num) {
		int idx;
		int i;
		int temp = channels*outer_idx*inter_num;
		idx = outer_idx*inter_num+inter_idx;
		sum_[idx] = input[temp];
		for(i = 1; i < channels; ++i){
			sum_[idx] += input[temp+i];

		}
	}
}

__global__ void kernel_div(float* input ,int inter_num, int outer_num, int channels, float* sum_){
	int id = blockIdx.x*blockDim.x+threadIdx.x;
	int inter_idx = id%inter_num;
	int outer_idx = id/inter_num;
	if(inter_idx < inter_num && outer_idx < outer_num) {
		int idx;
		int i;
		int temp = channels*outer_idx*inter_num;
		idx = outer_idx*inter_num+inter_idx;
		for(i = 0; i < channels; ++i){
			input[temp+i] /= sum_[idx];

		}
	}
}

void forward_softmax(struct LayerParameter* layer_parameter){
	Doutput_info();
	int outer_num, inter_num, channels;
	outer_num = layer_parameter->meds[0]->c;
	inter_num = layer_parameter->meds[0]->n;
	channels =  layer_parameter->bottoms[0]->count/(outer_num*inter_num);
	float* bottom_data = layer_parameter->bottoms[0]->gpu_data;
	float* top_data = layer_parameter->tops[0]->gpu_data;

	copy(bottom_data, top_data, layer_parameter->bottoms[0]->count);
//	dim3 block1((inter_num+CUDA_NUM_THREADS-1)/CUDA_NUM_THREADS,(outer_num+CUDA_NUM_THREADS-1)/CUDA_NUM_THREADS);
//	printf("softmax dimension %d, %d, %d, %d\n", layer_parameter->bottoms[0]->n, layer_parameter->bottoms[0]->c, layer_parameter->bottoms[0]->h, layer_parameter->bottoms[0]->w);
//	printf("inter_num %d outer_num %d\n", inter_num, outer_num);
//	printf("block x %d block y %d\n", (inter_num+CUDA_NUM_THREADS-1)/CUDA_NUM_THREADS, (outer_num+CUDA_NUM_THREADS-1)/CUDA_NUM_THREADS);
////for(;;);
//	kernel_max<<<1, block1>>>(bottom_data, inter_num, outer_num, channels, layer_parameter->meds[0]->gpu_data);
//	kernel_minus_max<<<1, block1>>>(top_data, inter_num, outer_num, channels, layer_parameter->meds[0]->gpu_data);
//	exp_gpu(top_data, top_data, layer_parameter->bottoms[0]->count);
//	kernel_sum<<<1, block1>>>(top_data, inter_num, outer_num, channels, layer_parameter->meds[0]->gpu_data);
//	kernel_div<<<1, block1>>>(top_data, inter_num, outer_num, channels, layer_parameter->meds[0]->gpu_data);
	kernel_max<<<(inter_num*outer_num+CUDA_NUM_THREADS-1)/CUDA_NUM_THREADS, CUDA_NUM_THREADS>>>(bottom_data, inter_num, outer_num, channels, layer_parameter->meds[0]->gpu_data);
	kernel_minus_max<<<(inter_num*outer_num+CUDA_NUM_THREADS-1)/CUDA_NUM_THREADS, CUDA_NUM_THREADS>>>(top_data, inter_num, outer_num, channels, layer_parameter->meds[0]->gpu_data);
	exp_gpu(top_data, top_data, layer_parameter->bottoms[0]->count);
	kernel_sum<<<(inter_num*outer_num+CUDA_NUM_THREADS-1)/CUDA_NUM_THREADS, CUDA_NUM_THREADS>>>(top_data, inter_num, outer_num, channels, layer_parameter->meds[0]->gpu_data);
	kernel_div<<<(inter_num*outer_num+CUDA_NUM_THREADS-1)/CUDA_NUM_THREADS, CUDA_NUM_THREADS>>>(top_data, inter_num, outer_num, channels, layer_parameter->meds[0]->gpu_data);
}
