#include "hip/hip_runtime.h"
#include "DL/layers/concat_layer.h"

void setup_concat(struct LayerParameter* layer_parameter){
	int bottom_size = layer_parameter->bottom_num;
	int shape[4] = {layer_parameter->bottoms[0]->n, layer_parameter->bottoms[0]->c, layer_parameter->bottoms[0]->h, layer_parameter->bottoms[0]->w};
	int axis = layer_parameter->parameter.concat_param.axis;
	int i;
	if(axis == 0){
		for (i = 1; i < bottom_size; ++i){
			shape[0] += layer_parameter->bottoms[i]->n;
		}
	}else{
		if(axis == 1){
			for (i = 1; i < bottom_size; ++i){
				shape[1] += layer_parameter->bottoms[i]->c;
			}
		}else{
			if(axis == 2){
				for (i = 1; i < bottom_size; ++i){
					shape[2] += layer_parameter->bottoms[i]->h;
				}
			}else{
				if(axis == 3){
					for (i = 1; i < bottom_size; ++i){
						shape[3] += layer_parameter->bottoms[i]->w;
					}
				}
			}
		}
	}
	MakeBlob(shape[0], shape[1], shape[2], shape[3], layer_parameter->tops[0]);
	Doutput_shape_info();
}
//input 的维度是outer_num*inter_num　根据这两个维度以及线程的维度就可以确定输入的维度分别为(blockIdx.x,threadIdx.x)
//output 的维度是outer_num*sum_inter_num，
//现在确定将input放在哪个位置位置为(blockIdx.x,threadIdx.x+n_sum_inter_num)
__global__ void concat_kernel(float* input, float* output,
		int outer_num, int inter_num, int sum_inter_num, int n_sum_inter_num){
	int id = blockDim.x*blockIdx.x+threadIdx.x;
	//int idx = blockIdx.x;
	//int idy = threadIdx.x;
	int idx = id/inter_num;
	int idy = id%inter_num;
	if(idx < outer_num && idy < inter_num){
		//int input_idx = idx*inter_num+idy;
		int output_idx = idx*sum_inter_num+idy+n_sum_inter_num;
		output[output_idx] = input[id];
	}
}

void forward_concat(struct LayerParameter* layer_parameter){
	Doutput_info();
//	int axis = layer_parameter->parameter.concat_param.axis;
//	int shape[4] = {layer_parameter->tops[0]->n, layer_parameter->tops[0]->c, layer_parameter->tops[0]->h, layer_parameter->tops[0]->w};
//	int outer_num=1, inter_num=1;
//	int i,j,k;
//	int bottom_size = layer_parameter->bottom_num;
//	float* top_data = layer_parameter->tops[0]->cpu_data;
//	int temp, temp1;
//	float* bottom_data;
//	for(i = 0; i < axis; ++i)
//		outer_num *= shape[i];
//	for(i = axis+1; i < 4; ++i)
//		inter_num *= shape[i];
//	for(i = 0; i < outer_num; ++i){
//		for(j = 0; j < bottom_size; ++j){
//			if(axis == 0){
//				temp = layer_parameter->bottoms[j]->n*i*inter_num;
//				temp1 = layer_parameter->bottoms[j]->n*inter_num;
//			}else{
//				if(axis == 1){
//					temp = layer_parameter->bottoms[j]->c*i*inter_num;
//					temp1 = layer_parameter->bottoms[j]->c*inter_num;
//				}else{
//					if(axis == 2){
//						temp = layer_parameter->bottoms[j]->h*i*inter_num;
//						temp1 = layer_parameter->bottoms[j]->h*inter_num;
//					}else{
//						if(axis ==3){
//							temp = layer_parameter->bottoms[j]->w*i*inter_num;
//							temp1 = layer_parameter->bottoms[j]->w*inter_num;
//						}
//					}
//				}
//			}
//			bottom_data = layer_parameter->bottoms[j]->cpu_data+temp;
//			for(k = 0; k < temp1; ++k){
//				top_data[0] = bottom_data[k];
//				++top_data;
//			}
//		}
//	}
	int i;
	int bottom_size = layer_parameter->bottom_num;
	int axis = layer_parameter->parameter.concat_param.axis;
	int shape[4] = {layer_parameter->tops[0]->n, layer_parameter->tops[0]->c, layer_parameter->tops[0]->h, layer_parameter->tops[0]->w};
	int bottom_shape[4];
	int outer_num=1;
	int inter_num=1;
	for(i = 0; i < axis; ++i)
		outer_num *= shape[i];
	for(i = axis+1; i < 4; ++i)
		inter_num *= shape[i];
	int n_sum_inter_num=0;
	int sum_inter_num = shape[axis]*inter_num;
	for(i = 0; i < bottom_size; ++i){
		bottom_shape[0] = layer_parameter->bottoms[i]->n;
		bottom_shape[1] = layer_parameter->bottoms[i]->c;
		bottom_shape[2] = layer_parameter->bottoms[i]->h;
		bottom_shape[3] = layer_parameter->bottoms[i]->w;
		inter_num*=bottom_shape[axis];
		//concat_kernel<<<(outer_num+CUDA_NUM_THREADS-1)/CUDA_NUM_THREADS, (inter_num+CUDA_NUM_THREADS-1)/CUDA_NUM_THREADS>>>(layer_parameter->bottoms[i]->gpu_data,
		//		layer_parameter->tops[0]->gpu_data, outer_num, inter_num, sum_inter_num, n_sum_inter_num);
		concat_kernel<<<(outer_num*inter_num+CUDA_NUM_THREADS-1)/CUDA_NUM_THREADS, CUDA_NUM_THREADS>>>(layer_parameter->bottoms[i]->gpu_data,
						layer_parameter->tops[0]->gpu_data, outer_num, inter_num, sum_inter_num, n_sum_inter_num);
		n_sum_inter_num+=inter_num;
		inter_num/=bottom_shape[axis];
//		hipError_t cudaerror = hipPeekAtLastError();
//		if(cudaerror == hipSuccess){
//			printf("success\n");
//		}else{
//			printf("%s \n", hipGetErrorString(cudaerror));
//		}
	}
//		hipError_t cudaerror = hipPeekAtLastError();
//		if(cudaerror == hipSuccess){
//			printf("success\n");
//		}else{
//			printf("%s \n", hipGetErrorString(cudaerror));
//		}
//		for(;;);
}
