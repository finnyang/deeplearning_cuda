#include "hip/hip_runtime.h"
#include "DL/layers/relu_layer.h"
#include <hip/hip_runtime.h>

__global__ void gpu_relu(float* input, float* output, int count, float negative_slope){
	int idx = blockDim.x*blockIdx.x+threadIdx.x;
	if(idx < count){
		output[idx] = input[idx] > 0? input[idx]:input[idx]*negative_slope;
	}
}

void setup_relu(struct LayerParameter* layer_parameter){
	layer_parameter->tops[0]->n = layer_parameter->bottoms[0]->n;
	layer_parameter->tops[0]->c = layer_parameter->bottoms[0]->c;
	layer_parameter->tops[0]->h = layer_parameter->bottoms[0]->h;
	layer_parameter->tops[0]->w = layer_parameter->bottoms[0]->w;
	layer_parameter->tops[0]->gpu_data = layer_parameter->bottoms[0]->gpu_data;
	layer_parameter->tops[0]->count = layer_parameter->bottoms[0]->count;
	layer_parameter->tops[0]->own = 0;
	Doutput_shape_info();
}
void forward_relu(struct LayerParameter* layer_parameter){
	Doutput_info();
	int count = layer_parameter->bottoms[0]->count;
	gpu_relu<<<(count+CUDA_NUM_THREADS-1)/CUDA_NUM_THREADS, CUDA_NUM_THREADS>>>(layer_parameter->bottoms[0]->gpu_data, layer_parameter->tops[0]->gpu_data,
			count, layer_parameter->parameter.relu_param.negative_slope);
}
