#include "DL/layers/innerproduct_layer.h"
#include <cblas.h>
#include <malloc.h>
#include <stdlib.h>
#include "DL/util/common_function.h"

void setup_innerproduct(struct LayerParameter* layer_parameter){
	struct InnerproductParameter innerproduct_param = layer_parameter->parameter.innerproduct_param;
	int num_out = innerproduct_param.num_output;
	struct Blob* output = layer_parameter->tops[0];
	struct Blob* input = layer_parameter->bottoms[0];
	int num_in = input->count/input->n;
	MakeBlob(input->n,num_out, 1, 1, output);

	struct Blob* weights = layer_parameter->learn_parameter[0];
	struct Blob* bias = layer_parameter->learn_parameter[1];
	MakeBlob(num_out, num_in, 1, 1, weights);
	MakeBlob(num_out, 1, 1, 1, bias);
	MakeBlob(input->n, 1, 1, 1, layer_parameter->meds[0]);
	set(layer_parameter->meds[0]->gpu_data, input->n, 1.0);
	Doutput_shape_info();
}

void forward_innerproduct(struct LayerParameter* layer_parameter){
	Doutput_info();
	struct Blob* input = layer_parameter->bottoms[0];
	struct Blob* output = layer_parameter->tops[0];
	struct Blob* weights = layer_parameter->learn_parameter[0];
	struct Blob* bias = layer_parameter->learn_parameter[1];
	int num_in = input->count/input->n;
	int num_out = output->c;
	int batch = input->n;
	struct Blob temp;
	MakeBlob(batch, 1, 1, 1, &temp);
	set(temp.gpu_data, batch, 1.0);
	const float a = 1.0;
	const float b = 0.0;
	hipblasSgemm(*(layer_parameter->p_cublas_handle), HIPBLAS_OP_T, HIPBLAS_OP_N, num_out, batch, num_in,
					&a, weights->gpu_data, num_in, input->gpu_data, num_in, &b, output->gpu_data, num_out);
	hipblasSgemm(*(layer_parameter->p_cublas_handle), HIPBLAS_OP_N, HIPBLAS_OP_N, num_out, batch, 1,
					&a, bias->gpu_data, num_out, layer_parameter->meds[0]->gpu_data, 1, &a, output->gpu_data, num_out);
//	hipblasSgemm(*(layer_parameter->p_cublas_handle), HIPBLAS_OP_N, HIPBLAS_OP_N, num_out, batch, 1,
//						&a, bias->gpu_data, num_out, temp.gpu_data, 1, &a, output->gpu_data, num_out);
	FreeBlob(&temp);
}
